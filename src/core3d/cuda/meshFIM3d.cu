#include "hip/hip_runtime.h"

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          modified to use only 7 floats for triMem
//1. #define TRIMEMLENGTH   7
//2. in FIMCuda and run_neighbor_check, add initilize old at the begining of iteration
//3. in FIMCuda and run_neighbor_check, s_triMem[tx*TRIMEMLENGTH + 3 + C] = TC after each iteration instead of s_triMem[tx*TRIMEMLENGTH + 6 + C] = TC
//4. in FIMCuda and run_neighbor_check, in the reconcile step, there should be no +3 in fetching the location of triMem
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "meshFIM3d.h"
#include "Vec.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#ifdef WIN32
#include <io.h>
#define unlink _unlink
#else
#include <unistd.h>
#endif
#include "CUDADefines.h"
#include <time.h>
#include <mycutil.h>
extern "C" {
#include <metis.h>
}


/////declaration for cuda kernels///////////////////////////
extern __global__ void run_reduction(bool *con, bool *blockCon, int* ActiveList, int nActiveBlock, int* blockSizes);


extern __global__ void FIMCuda(float3* d_tetMem0, float3* d_tetMem1, float4* d_tetT, float* d_vertT, float* d_speedInv, int* d_vertMem, int* d_vertMemOutside,
    int* d_BlockSizes, bool* d_con, int* d_ActiveList,
    int m_maxNumInVert, int m_maxVertMappingInside, int m_maxNumOutVertMapping, int nIter);

extern __global__ void CopyOutBack(float4* d_tetT, float* d_vertT, int* d_vertMem, int* d_vertMemOutside, int* d_BlockSizes, int* d_ActiveList, int m_maxNumInVert, int m_maxNumTotalTets, int m_maxVertMappingInside, int m_maxVertMappingOutside);


extern __global__ void run_check_neghbor(float3* d_tetMem0, float3* d_tetMem1, float4* d_tetT, float* d_speedInv, int* d_vertMem, int* d_vertMemOutside,
    int* d_BlockSizes, bool* d_con, int* d_ActiveList,
    int m_maxNumInVert, int m_maxVertMappingInside, int m_maxNumOutVertMapping);

#if __DEVICE_EMULATION__

bool InitCUDA(bool verbose = false)
{
  return true;
}


#else

bool InitCUDA(bool verbose = false)
{
  int count = 0;
  int i = 0;

  hipGetDeviceCount(&count);
  if(count == 0)
  {
    fprintf(stderr, "There is no device.\n");
    return false;
  }

  for(i = 0; i < count; i++)
  {
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop, i) == hipSuccess)
    {
      if(prop.major >= 1)
      {
        break;
      }
    }
  }
  if(i == count)
  {
    fprintf(stderr, "There is no device supporting CUDA.\n");
    return false;
  }

  hipDeviceProp_t props;
  cudaSafeCall(hipSetDevice(0));

  cudaSafeCall(hipGetDeviceProperties(&props, 0));

  if (verbose) {
    printf("Device 0: \"%s\" with Compute %d.%d capability\n",  props.name, props.major, props.minor);

    printf("CUDA initialized.\n");
  }
  return true;
}

#endif

/////////////////////////////////////////////////////////////////////////////
//create .mesh file from trimesh faces and call partnmesh function
//to partition and create intermediate mesh.npart.N file and then read this file
void meshFIM3d::GraphPartition_METIS2(int& numBlock, int maxNumBlockVerts, bool verbose)
{

  FILE * outf;

  outf = fopen("tmp.mesh", "w+");
  if(outf == NULL)
  {
    printf("Cannot open mesh file to write!!!!\n");
    exit(1);
  }
  int sz = m_meshPtr->tets.size();

  fprintf(outf, "%d 2\n", sz);

  for(int i = 0; i < sz; i++)
    fprintf(outf, "%d %d %d %d\n", m_meshPtr->tets[i].v[0] + 1, m_meshPtr->tets[i].v[1] + 1, m_meshPtr->tets[i].v[2] + 1, m_meshPtr->tets[i].v[3] + 1);

  fclose(outf);


  int numVert = m_meshPtr->vertices.size();

  m_PartitionLabel.resize(numVert);

  char outputFileName[512];

  char meshfile[] = "tmp.mesh";

  if(numBlock == 0)
  {
    numBlock = MAX(numVert / maxNumBlockVerts - 10, 1);


    do
    {
      numBlock++;

      m_BlockSizes.resize(numBlock);
      for(int i = 0; i < numBlock; i++)
      {
        m_BlockSizes[i] = 0;
      }
      partnmesh(meshfile,numBlock,verbose?1:0);

      sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);


      FILE* partFile = fopen(outputFileName, "r+");
      if(partFile == NULL)
      {
        printf("NO part file found!!!!\n");
        exit(1);
      }
      for(int i = 0; i < numVert; i++)
      {
        fscanf(partFile, "%d", &m_PartitionLabel[i]);
      }

      for(int i = 0; i < numVert; i++)
      {
        m_BlockSizes[m_PartitionLabel[i]]++;
      }
      m_maxNumInVert = 0;

      for(int i = 0; i < numBlock; i++)
      {

        m_maxNumInVert = MAX(m_maxNumInVert, m_BlockSizes[i]);
      }

      fclose(partFile);

      sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
      unlink(outputFileName);
      sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
      unlink(outputFileName);

    }
    while(m_maxNumInVert != maxNumBlockVerts);
  }
  else
  {
    m_BlockSizes.resize(numBlock);
    for(int i = 0; i < numBlock; i++)
    {
      m_BlockSizes[i] = 0;
    }

    partnmesh(meshfile, numBlock,verbose?1:0);

    sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);

    FILE* partFile = fopen(outputFileName, "r+");
    if(partFile == NULL)
    {
      printf("NO part file found!!!!\n");
      exit(1);
    }

    for(int i = 0; i < numVert; i++)
    {
      fscanf(partFile, "%d", &m_PartitionLabel[i]);
    }

    for(int i = 0; i < numVert; i++)
    {
      m_BlockSizes[m_PartitionLabel[i]]++;
    }
    m_maxNumInVert = 0;

    for(int i = 0; i < numBlock; i++)
    {
      m_maxNumInVert = MAX(m_maxNumInVert, m_BlockSizes[i]);
    }

    if (verbose)
      printf("max num vert is : %d\n", m_maxNumInVert);
    fclose(partFile);

    sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
    unlink(outputFileName);
    sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
    unlink(outputFileName);
  }

  srand((unsigned)time(NULL));

  if (verbose)
    printf("numBlock is : %d\n", numBlock);

  m_PartitionInVerts.resize(numBlock);

  for(int i = 0; i < numVert; i++)
  {
    m_PartitionInVerts[m_PartitionLabel[i]].push_back(i);
  }
  unlink("tmp.mesh");
}

void meshFIM3d::GraphPartition_Square(int squareLength, int squareWidth, int squareHeight, int blockLength, int blockWidth, int blockHeight, bool verbose)
{
  int numVert = m_meshPtr->vertices.size();
  m_PartitionLabel.resize(numVert);

  int numBlockLength = (squareLength / blockLength);
  int numBlockWidth = (squareWidth / blockWidth);
  int numBlockHeight = (squareHeight / blockHeight);
  int numBlock = numBlockLength * numBlockWidth*numBlockHeight;


  for(int k = 0; k < squareHeight; k++)
    for(int i = 0; i < squareWidth; i++)
      for(int j = 0; j < squareLength; j++)
      {

        int index = k * squareLength * squareWidth + i * squareLength + j;
        int k2 = k;
        int i2 = i;
        int j2 = j;
        m_PartitionLabel[index] = (k2 / blockHeight) * numBlockLength * numBlockWidth + (i2 / blockWidth) * numBlockLength + (j2 / blockLength);
      }

  m_BlockSizes.resize(numBlock);

  for(int i = 0; i < numBlock; i++)
    m_BlockSizes[i] = 0;

  m_PartitionInVerts.resize(numBlock);

  for(int i = 0; i < numVert; i++)
  {
    m_PartitionInVerts[m_PartitionLabel[i]].push_back(i);
    m_BlockSizes[m_PartitionLabel[i]]++;
  }

  m_maxNumInVert = 0;

  for(int i = 0; i < numBlock; i++)
  {
    m_maxNumInVert = MAX(m_maxNumInVert, m_BlockSizes[i]);
  }
  if (verbose)
    printf("final number of blocks: %d\n", numBlock);
}

std::vector < std::vector < float > >  meshFIM3d::GenerateData(size_t maxIters, bool verbose)
{
  int numVert = m_meshPtr->vertices.size();

  if(!InitCUDA(verbose))
  {
    exit(1);
  }

  float* h_tetMem0;
  float* h_tetMem1;
  float* h_tetT;
  float* h_vertT;
  int* h_vertMem;
  int* h_vertMemOutside;
  bool* h_blockCon;
  int* h_BlockSizes;
  int* h_BlockLabel;
  vector<int> h_ActiveList;
  vector<int> h_ActiveListNew;

  int* d_ActiveList = 0;
  bool* d_con;
  bool* d_blockCon;
  float3* d_tetMem0;
  float3* d_tetMem1;
  float4* d_tetT;
  float* d_vertT;
  float* d_speedInv;
  int* d_vertMem;
  int* d_vertMemOutside;
  int* d_BlockSizes;

  GetTetMem(h_tetMem0, h_tetMem1, h_tetT);
  GetVertMem(h_vertMem, h_vertMemOutside);
  h_vertT = (float*)malloc(sizeof(float)* m_maxNumInVert * m_numBlock);


  h_blockCon = (bool*)malloc(sizeof(bool) * m_numBlock);
  h_BlockLabel = (int*)malloc(sizeof(int)* m_numBlock);
  h_BlockSizes = (int*)malloc(sizeof(int)* m_numBlock);
  memset(h_blockCon, 1, sizeof(bool) * m_numBlock);
  for(int i = 0; i < m_numBlock; i++) {
    h_BlockLabel[i] = FARP;
    h_BlockSizes[i] = m_BlockSizes[i];
  }

  ////////////////////initialize the seed points for h_tetT//////////////////////////

  if (verbose)
    printf("Seed size is %d, source block is %d\n", m_SeedPoints.size(),
        m_PartitionLabel.empty()?-1:
        (m_PartitionLabel[m_SeedPoints.empty()?0:m_SeedPoints[0]]));
  for(int i = 0; i < m_SeedPoints.size(); i++)
  {
    int seed = m_SeedPoints[i];
    int seedBelongToBlock = m_PartitionLabel[seed];
    m_ActiveBlocks.insert(m_ActiveBlocks.end(), seedBelongToBlock);
    h_blockCon[seedBelongToBlock] = false;
    h_BlockLabel[seedBelongToBlock] = ACTIVE;
    for(int j = 0; j < m_blockVertMapping[seed].size(); j++)
    {
      h_tetT[m_blockVertMapping[seed][j]] = 0.0;
    }
  }

  int numActive = m_ActiveBlocks.size();

  if (verbose)
    printf("Active block number is %d.\n", numActive);


  h_ActiveList.resize(m_numBlock);

  set<int>::iterator activeiter = m_ActiveBlocks.begin();
  for(int i = 0; activeiter != m_ActiveBlocks.end(); activeiter++)
    h_ActiveList[i++] = *activeiter;


  unsigned int timerstart, timerend = 0;


  ///////////////////////malloc GPU memory/////////////////////////////////
  cudaSafeCall((hipMalloc((void**)&d_con, sizeof(bool) * m_numBlock * m_maxNumInVert)));
  cudaSafeCall((hipMalloc((void**)&d_tetMem0, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_tetMem1, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_tetT, sizeof(float)* 4 * m_maxNumTotalTets * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_vertT, sizeof(float)* m_maxNumInVert * m_numBlock)));
  cudaSafeCall( hipMalloc( (void**) &d_speedInv, sizeof(float) * m_maxNumTotalTets * m_numBlock) );
  cudaSafeCall((hipMalloc((void**)&d_vertMem, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingInside)));
  cudaSafeCall((hipMalloc((void**)&d_vertMemOutside, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingOutside)));
  cudaSafeCall((hipMalloc((void**)&d_BlockSizes, sizeof(int)* m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_blockCon, sizeof(bool) * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_ActiveList, sizeof(int)* m_numBlock)));


  //////////////////copy to gpu memories///////////////////////////////
  cudaSafeCall((hipMemcpy(d_tetMem0, h_tetMem0, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_tetMem1, h_tetMem1, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_tetT, h_tetT, sizeof(float)* 4 * m_maxNumTotalTets * m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_vertMem, h_vertMem, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingInside, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_vertMemOutside, h_vertMemOutside, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingOutside, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_BlockSizes, h_BlockSizes, sizeof(int)* m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_blockCon, h_blockCon, sizeof(bool) * m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemset(d_vertT, 0, sizeof(float)* m_maxNumInVert * m_numBlock)));

  int nTotalIter = 0;
  int nIter = 2;


  hipFuncSetCacheConfig(reinterpret_cast<const void*>(FIMCuda), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(run_check_neghbor), hipFuncCachePreferShared);

  vector< vector<float> > tmp_h_verrT;
  vector< vector<float> > tmp_h_verrT2;
  tmp_h_verrT.resize(m_numBlock);
  tmp_h_verrT2.resize(m_numBlock);

  int totalIterationNumber = 0;
  timerstart = clock();

  //the result vector
  std::vector< std::vector < float > > result;
  m_meshPtr->vertT.resize(1);
  m_meshPtr->vertT[0].resize(numVert);

  int maxActive = 0;
  while(numActive > 0)
  {
    maxActive = MAX(maxActive, numActive);
    ///////step 1: run solver /////////////////////////////////////
    nTotalIter++;
    //don't do more than maxIters
    if (nTotalIter > maxIters) break;
    if (verbose ) {
      size_t act = numActive / 3;
      for(size_t ab = 0; ab < 60; ab++) {
        if (ab < act)
          printf("=");
        else
          printf(" ");
      }
      printf(" %d Active blocks.\n", numActive);
    }
    totalIterationNumber += numActive;
    dim3 dimGrid(numActive, 1);
    dim3 dimBlock(m_maxNumTotalTets, 1);
    cudaSafeCall(hipMemcpy(d_ActiveList, &h_ActiveList[0], sizeof(int)* m_numBlock, hipMemcpyHostToDevice));
    int sharedSize = sizeof(float)* 4 * m_maxNumTotalTets + sizeof(int)* m_maxNumInVert * m_maxVertMappingInside;
    cudaSafeCall((FIMCuda << <dimGrid, dimBlock, sharedSize >> >(d_tetMem0, d_tetMem1, d_tetT, d_vertT, d_speedInv, d_vertMem, d_vertMemOutside,
            d_BlockSizes, d_con, d_ActiveList, m_maxNumInVert, m_maxVertMappingInside, m_maxVertMappingOutside, nIter)));

    dimBlock = dim3(m_maxNumInVert, 1);
    CopyOutBack << <dimGrid, dimBlock >> >(d_tetT, d_vertT, d_vertMem, d_vertMemOutside, d_BlockSizes, d_ActiveList, m_maxNumInVert, m_maxNumTotalTets, m_maxVertMappingInside, m_maxVertMappingOutside);
    dimBlock = dim3(m_maxNumInVert, 1);
    run_reduction << <dimGrid, dimBlock >> >(d_con, d_blockCon, d_ActiveList, numActive, d_BlockSizes);
    cudaSafeCall(hipMemcpy(h_blockCon, d_blockCon, m_numBlock * sizeof(bool), hipMemcpyDeviceToHost));
    int nOldActiveBlock = numActive;
    numActive = 0;
    h_ActiveListNew.clear();
    for(uint i = 0; i < nOldActiveBlock; i++)
    {
      uint currBlkIdx = h_ActiveList[i];
      h_BlockLabel[currBlkIdx] = FARP;
      if(!h_blockCon[currBlkIdx]) // if not converged
      {
        h_BlockLabel[currBlkIdx] = ACTIVE;
        //h_ActiveList[numActive++] = currBlkIdx;
      }
    }

    for(uint i = 0; i < nOldActiveBlock; i++)
    {
      // check neighbors of current active tile
      uint currBlkIdx = h_ActiveList[i];

      if(h_blockCon[currBlkIdx]) //converged
      {
        //h_BlockLabel[currBlkIdx] == FARP;
        set<int> nb = m_BlockNeighbor[currBlkIdx];

        set<int>::iterator iter;
        for(iter = nb.begin(); iter != nb.end(); iter++)
        {
          int currIdx = *iter;

          if(h_BlockLabel[currIdx] == FARP)
          {
            h_BlockLabel[currIdx] = ACTIVE;
            h_ActiveListNew.push_back(currIdx);
          }
        }
      }
    }

    for(uint i = 0; i < nOldActiveBlock; i++)
    {
      uint currBlkIdx = h_ActiveList[i];
      if(!h_blockCon[currBlkIdx]) // if not converged
      {
        h_ActiveList[numActive++] = currBlkIdx;
      }
    }
    //////////////////////////////////////////////////////////////////
    // 4. run solver only once for neighbor blocks of converged block
    // current active list contains active blocks and neighbor blocks of
    // any converged blocks
    if(h_ActiveListNew.size() > 0)
    {

      int numActiveNew = h_ActiveListNew.size();

      cudaSafeCall(hipMemcpy(d_ActiveList, &h_ActiveListNew[0], numActiveNew * sizeof(int), hipMemcpyHostToDevice));
      dim3 dimGrid(numActiveNew, 1);
      dim3 dimBlock(m_maxNumTotalTets, 1);

      int sharedSize = sizeof(float4) * m_maxNumTotalTets + sizeof(int)* m_maxNumInVert * m_maxVertMappingInside;
      run_check_neghbor << <dimGrid, dimBlock, sharedSize >> >(d_tetMem0, d_tetMem1, d_tetT, d_speedInv, d_vertMem, d_vertMemOutside,
          d_BlockSizes, d_con, d_ActiveList, m_maxNumInVert, m_maxVertMappingInside, m_maxVertMappingOutside);

      ////////////////////////////////////////////////////////////////
      // 5. reduction
      ///////////////////////////////////////////////////////////////
      dimGrid = dim3(numActiveNew, 1);
      dimBlock = dim3(m_maxNumInVert, 1);
      run_reduction << <dimGrid, dimBlock >> >(d_con, d_blockCon, d_ActiveList, numActiveNew, d_BlockSizes);

      //////////////////////////////////////////////////////////////////
      // 6. update active list
      // read back active volume from the device and add
      // active block to active list on the host memory
      cudaSafeCall(hipMemcpy(h_blockCon, d_blockCon, m_numBlock * sizeof(bool), hipMemcpyDeviceToHost));
      for(uint i = 0; i < h_ActiveListNew.size(); i++)
      {

        uint currBlkIdx = h_ActiveListNew[i];
        if(!h_blockCon[currBlkIdx]) // false : activate block (not converged)
        {
          h_ActiveList[numActive++] = currBlkIdx;
        }
        else h_BlockLabel[currBlkIdx] = FARP;
      }
    }
    ////////////////////////copy values from each iteration
    cudaSafeCall(hipMemcpy(h_vertT, d_vertT,
          sizeof(float)* m_maxNumInVert * m_numBlock, hipMemcpyDeviceToHost));
    for(int i = 0; i < m_numBlock; i++)
    {
      for(int j = 0; j < m_PartitionInVerts[i].size(); j++)
      {
        m_meshPtr->vertT[0][m_PartitionInVerts[i][j]] =
          h_vertT[i * m_maxNumInVert + j];
      }
    }
    result.push_back(m_meshPtr->vertT[0]);
    ////////////////////////////////END copy
  }
  cudaSafeCall(hipDeviceSynchronize());
  timerend = clock();
  double duration = (double)(timerend - timerstart) / CLOCKS_PER_SEC;

  if (verbose)
    printf("Computing time : %.10lf s\n",duration);

  cudaSafeCall(hipDeviceSynchronize());

  if (verbose)
    printf("num of max active %d\n", maxActive);

  if (verbose) {
    printf("The iteration number: %d\n", nTotalIter);
    printf("The total iteration number: %d\n", totalIterationNumber);
  }
  cudaSafeCall(hipFree(d_con));
  cudaSafeCall(hipFree(d_blockCon));
  cudaSafeCall(hipFree(d_BlockSizes));

  free(h_blockCon);
  free(h_BlockSizes);
  return result;
}

void meshFIM3d::PartitionTets(int numBlock, bool verbose)
{
  ///////////////////////////////////step 3: partition faces//////////////////////////////////////
  if (verbose)
    printf("Start PartitionTets ...");
  m_PartitionTets.resize(numBlock);
  m_PartitionNbTets.resize(numBlock);

  int numTets = m_meshPtr->tets.size();
  int numVerts = m_meshPtr->vertices.size();
  TetMesh::Tet t;

  vector<TetMesh::Tet> virtualTets;
  vector<int> virtualTetCnt;

  virtualTetCnt.resize(numBlock);
  m_PartitionVirtualTets.resize(numBlock);
  set<int> labels;

  for(int i = 0; i < numTets; i++)
  {
    t = m_meshPtr->tets[i];
    int vfCnt = m_meshPtr->tetVirtualTets[i].size();


    int obtusevert = t.obtuseV;
    if(obtusevert >= 0)
    {
      virtualTetCnt[m_PartitionLabel[t[obtusevert]]] += vfCnt;
      m_PartitionVirtualTets[m_PartitionLabel[t[obtusevert]]].insert(m_PartitionVirtualTets[m_PartitionLabel[t[obtusevert]]].end(), m_meshPtr->tetVirtualTets[i].begin(), m_meshPtr->tetVirtualTets[i].end());
    }
    labels.clear();
    for(int m = 0; m < 4; m++)
      labels.insert(labels.begin(), m_PartitionLabel[t[m]]);
    if(labels.size() == 1)
    {
      m_PartitionTets[*(labels.begin())].push_back(i);
    }
    else if(labels.size() > 1)
    {
      set<int>::iterator it = labels.begin();
      for(set<int>::iterator it = labels.begin(); it != labels.end(); it++)
      {
        m_PartitionNbTets[*it].push_back(i);
      }
    }
    else
      printf("Error!!\n");
  }

  vector<int> PartitionToltalTets;
  PartitionToltalTets.resize(numBlock);
  m_maxNumTotalTets = 0;
  for(int j = 0; j < numBlock; j++)
  {
    PartitionToltalTets[j] = m_PartitionTets[j].size() + m_PartitionNbTets[j].size() + virtualTetCnt[j];
    m_maxNumTotalTets = MAX(PartitionToltalTets[j], m_maxNumTotalTets);
  }

  if (verbose)
    printf("m_maxNumTotalTets is %d\n", m_maxNumTotalTets);


  //calculate block neighbors.
  m_BlockNeighbor.resize(numBlock);
  for(int i = 0; i < numVerts; i++)
  {
    vector<int> nbs = m_meshPtr->neighbors[i];
    for(int j = 0; j < nbs.size(); j++)
    {
      int nb = nbs[j];
      if(m_PartitionLabel[nb] != m_PartitionLabel[i])
        m_BlockNeighbor[m_PartitionLabel[i]].insert(m_BlockNeighbor[m_PartitionLabel[i]].end(), m_PartitionLabel[nb]);
    }

  }
  if (verbose)
    printf("done!\n");
}

bool meshFIM3d::gettetmem(vector<float>& tetmem, TetMesh::Tet t)
{
  bool needswap = false;
  tetmem.resize(6);
  point A = m_meshPtr->vertices[t[0]];
  point B = m_meshPtr->vertices[t[1]];
  point C = m_meshPtr->vertices[t[2]];
  point D = m_meshPtr->vertices[t[3]];

  point AB = B - A;
  point AC = C - A;
  point AD = D - A;

  AC = C - A;
  AD = D - A;
  point BC = C - B;
  point CD = D - C;
  point BD = D - B;

  tetmem[0] = vMv(AC, t.M, BC);
  tetmem[1] = vMv(BC, t.M, CD);
  tetmem[2] = vMv(AC, t.M, CD);
  tetmem[3] = vMv(AD, t.M, BD);
  tetmem[4] = vMv(AC, t.M, AD);
  tetmem[5] = vMv(BC, t.M, BD);

  return needswap;

}

void meshFIM3d::GetTetMem(float* &h_tetMem0, float* &h_tetMem1, float* &h_tetT)
{
  h_tetMem0 = (float*)malloc(3 * sizeof(float)* m_maxNumTotalTets * m_numBlock);
  h_tetMem1 = (float*)malloc(3 * sizeof(float)* m_maxNumTotalTets * m_numBlock);
  h_tetT = (float*)malloc(4 * sizeof(float)* m_maxNumTotalTets * m_numBlock);

  int numTets = m_meshPtr->tets.size();

  int numVert = m_meshPtr->vertices.size();

  m_blockVertMapping.resize(numVert); //for each vertex, store the addresses where it appears in the global triMem array.

  TetMesh::Tet t;


  for(int i = 0; i < m_numBlock; i++)
  {
    int blockIdx = i * m_maxNumTotalTets * 3;
    int numPF = m_PartitionTets[i].size();
    for(int j = 0; j < numPF; j++)
    {

      t = m_meshPtr->tets[m_PartitionTets[i][j]];
      vector<float> tetmem;
      bool needswap = gettetmem(tetmem, t);

      h_tetMem0[blockIdx + j * 3 + 0] = tetmem[0];
      h_tetMem0[blockIdx + j * 3 + 1] = tetmem[1];
      h_tetMem0[blockIdx + j * 3 + 2] = tetmem[2];

      h_tetMem1[blockIdx + j * 3 + 0] = tetmem[3];
      h_tetMem1[blockIdx + j * 3 + 1] = tetmem[4];
      h_tetMem1[blockIdx + j * 3 + 2] = tetmem[5];

      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;

      m_blockVertMapping[t[0]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 0);
      m_blockVertMapping[t[3]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 3);

      if(needswap)
      {
        m_blockVertMapping[t[1]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
        m_blockVertMapping[t[2]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
      }
      else
      {
        m_blockVertMapping[t[1]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
        m_blockVertMapping[t[2]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);

      }
    }

  }

  for(int i = 0; i < m_numBlock; i++)
  {
    int blockIdx = i * m_maxNumTotalTets * 3;

    int numPF = m_PartitionTets[i].size();
    int numPNF = m_PartitionNbTets[i].size();
    int numPVF = m_PartitionVirtualTets[i].size();

    int k = 0;
    int l = 0;

    for(int j = numPF; j < m_maxNumTotalTets; j++)
    {

      if(j < numPF + numPNF)
      {

        vector<float> tetmem;
        t = m_meshPtr->tets[m_PartitionNbTets[i][k]];
        bool needswap = gettetmem(tetmem, t);

        h_tetMem0[blockIdx + j * 3 + 0] = tetmem[0];
        h_tetMem0[blockIdx + j * 3 + 1] = tetmem[1];
        h_tetMem0[blockIdx + j * 3 + 2] = tetmem[2];

        h_tetMem1[blockIdx + j * 3 + 0] = tetmem[3];
        h_tetMem1[blockIdx + j * 3 + 1] = tetmem[4];
        h_tetMem1[blockIdx + j * 3 + 2] = tetmem[5];

        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;

        m_blockVertMapping[t[0]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 0);
        m_blockVertMapping[t[3]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 3);
        if(needswap)
        {
          m_blockVertMapping[t[1]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
          m_blockVertMapping[t[2]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
        }
        else
        {
          m_blockVertMapping[t[1]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
          m_blockVertMapping[t[2]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
        }
        k++;
      }
      else if(j < numPF + numPNF + numPVF)
      {
        t = m_PartitionVirtualTets[i][l];
        vector<float> tetmem;
        bool needswap = gettetmem(tetmem, t);

        h_tetMem0[blockIdx + j * 3 + 0] = tetmem[0];
        h_tetMem0[blockIdx + j * 3 + 1] = tetmem[1];
        h_tetMem0[blockIdx + j * 3 + 2] = tetmem[2];

        h_tetMem1[blockIdx + j * 3 + 0] = tetmem[3];
        h_tetMem1[blockIdx + j * 3 + 1] = tetmem[4];
        h_tetMem1[blockIdx + j * 3 + 2] = tetmem[5];

        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;

        m_blockVertMapping[t[0]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 0);
        m_blockVertMapping[t[3]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 3);
        if(needswap)
        {
          m_blockVertMapping[t[1]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
          m_blockVertMapping[t[2]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
        }
        else
        {
          m_blockVertMapping[t[1]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
          m_blockVertMapping[t[2]%m_blockVertMapping.size()].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
        }
        l++;
      }
      else
      {
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;
      }
    }
  }
}

void meshFIM3d::GetVertMem(int* &h_vertMem, int* &h_vertMemOutside)
{

  int numVert = m_meshPtr->vertices.size();

  m_blockVertMappingInside.resize(numVert);
  m_blockVertMappingOutside.resize(numVert);

  m_maxNumVertMapping = 0;

  for(int i = 0; i < m_numBlock; i++)
  {
    int triIdx = i * TETMEMLENGTH * m_maxNumTotalTets;

    for(int m = 0; m < m_PartitionInVerts[i].size(); m++)
    {

      m_maxNumVertMapping = MAX(m_maxNumVertMapping, m_blockVertMapping[i%m_blockVertMapping.size()].size());

      vector<int> tmp = m_blockVertMapping[m_PartitionInVerts[i][m]%m_blockVertMapping.size()];


      for(int n = 0; n < tmp.size(); n++)
      {
        if(tmp[n] >= triIdx + 0 && tmp[n] < triIdx + m_maxNumTotalTets * TETMEMLENGTH)
          m_blockVertMappingInside[m_PartitionInVerts[i][m]].push_back(tmp[n]);
        else
        {
          m_blockVertMappingOutside[m_PartitionInVerts[i][m]].push_back(tmp[n]);
        }
      }
    }
  }

  m_maxVertMappingInside = 0;
  m_maxVertMappingOutside = 0;
  for(int i = 0; i < numVert; i++)
  {
    m_maxVertMappingInside = MAX(m_maxVertMappingInside, (m_blockVertMappingInside[i].size()));
    m_maxVertMappingOutside = MAX(m_maxVertMappingOutside, (m_blockVertMappingOutside[i].size()));
  }

  h_vertMem = (int*)malloc(sizeof(int)* m_maxVertMappingInside * m_maxNumInVert * m_numBlock);
  for(int i = 0; i < m_numBlock; i++)
  {
    int vertIdx = i * m_maxVertMappingInside * m_maxNumInVert;

    for(int m = 0; m < m_PartitionInVerts[i].size(); m++)
    {

      int tmpsize = m_blockVertMappingInside[m_PartitionInVerts[i][m]].size();

      int n = 0;
      for(; n < tmpsize; n++)
        h_vertMem[vertIdx + m * m_maxVertMappingInside + n] = m_blockVertMappingInside[m_PartitionInVerts[i][m]][n];
      for(; n < m_maxVertMappingInside; n++)
        h_vertMem[vertIdx + m * m_maxVertMappingInside + n] = -1 + i * m_maxNumTotalTets * TETMEMLENGTH;

    }

    for(int m = m_PartitionInVerts[i].size() * m_maxVertMappingInside; m < m_maxNumInVert * m_maxVertMappingInside; m++)
    {
      h_vertMem[vertIdx + m] = -1 + i * m_maxNumTotalTets*TETMEMLENGTH;
    }
  }


  h_vertMemOutside = (int*)malloc(m_maxNumInVert * m_numBlock * m_maxVertMappingOutside * sizeof(int));

  for(int i = 0; i < m_numBlock; i++)
  {
    int vertIdx = i * m_maxVertMappingOutside * m_maxNumInVert;

    for(int m = 0; m < m_PartitionInVerts[i].size(); m++)
    {

      int tmpsize = m_blockVertMappingOutside[m_PartitionInVerts[i][m]].size();

      int n = 0;
      for(; n < tmpsize; n++)
        h_vertMemOutside[vertIdx + m * m_maxVertMappingOutside + n] = m_blockVertMappingOutside[m_PartitionInVerts[i][m]][n];
      for(; n < m_maxVertMappingOutside; n++)
        h_vertMemOutside[vertIdx + m * m_maxVertMappingOutside + n] = -1;

    }

    for(int m = m_PartitionInVerts[i].size() * m_maxVertMappingOutside; m < m_maxNumInVert * m_maxVertMappingOutside; m++)
    {
      h_vertMemOutside[vertIdx + m] = -1;
    }
  }
}
