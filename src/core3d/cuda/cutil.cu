#include <cutil.h>
#include <cstdio>

void cudaSafeCall()
{
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
  {
    fprintf( stderr, "cudaSafeCall() failed at ");
    printFilenameAndLine();
    fprintf( stderr, " : %s\n", hipGetErrorString( err ) );
    exit( -1 );
  }
#endif

  return;
}

void cudaSafeCall(hipError_t err)
{
#ifdef CUDA_ERROR_CHECK
  if ( hipSuccess != err )
  {
    fprintf( stderr, "cudaSafeCall() failed at ");
    printFilenameAndLine();
    fprintf( stderr, " : %s\n", hipGetErrorString( err ) );
    exit( -1 );
  }
#endif

  return;
}
