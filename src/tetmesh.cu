#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "tetmesh.h"
//#include "Stdafx.h"

using namespace std;


void TetMesh::init(double* pointlist, int numpoint, int*trilist, int numtri, int* tetlist, int numtet, int numattr, double* attrlist)
{
	numVert = numpoint;
	numTet  = numtet;

	printf("Total num of vertices is %d\n", numVert);
	printf("Total num of tets is     %d\n", numTet);
	vertices.resize(numpoint);
	tets.resize(numtet);
	for(int i =0; i< numpoint; i++)
	{
		vertices[i][0] = pointlist[3*i+0];
		vertices[i][1] = pointlist[3*i+1];
		vertices[i][2] = pointlist[3*i+2];




	}

	//check the index start from 0 or 1
	int minidx = 1000000000;
	for(int i=0; i<numtet*4; i++)
	{
		minidx = MIN(minidx, tetlist[i]);

	}

	if(minidx == 0)
	{
		for(int i =0; i< numtet; i++)
		{

			tets[i][0] = tetlist[4*i+0];  
			tets[i][1] = tetlist[4*i+1];
			tets[i][2] = tetlist[4*i+2];
			tets[i][3] = tetlist[4*i+3];
			tets[i].obtuseV = -1;
			tets[i].M[0] = 1.0;
			tets[i].M[1] = 0.0;
			tets[i].M[2] = 0.0;
			tets[i].M[3] = 1.0;
			tets[i].M[4] = 0.0;
			tets[i].M[5] = 1.0;

		}
	}
	else if(minidx == 1)
	{
		for(int i =0; i< numtet; i++)
		{
			tets[i][0] = tetlist[4*i+0]-1;  // -1 because the oringal index is from 1 and change it to 0
			tets[i][1] = tetlist[4*i+1]-1;
			tets[i][2] = tetlist[4*i+2]-1;
			tets[i][3] = tetlist[4*i+3]-1;
			tets[i].obtuseV = -1;
			tets[i].M[0] = 1.0;
			tets[i].M[1] = 0.0;
			tets[i].M[2] = 0.0;
			tets[i].M[3] = 1.0;
			tets[i].M[4] = 0.0;
			tets[i].M[5] = 1.0;
		}
	}
	else
	{
		printf("error!!! index not start from 0 or 1!!\n");
	}


	if(numattr > 0)
	{
		for(int i =0; i< numtet; i++)
		{
			int mat = (int)attrlist[i];
			switch(mat)
			{
			case 0:
				tets[i].M[0] = 1.0;
				tets[i].M[1] = 0.0;
				tets[i].M[2] = 0.0;
				tets[i].M[3] = 1.0;
				tets[i].M[4] = 0.0;
				tets[i].M[5] = 1.0;
				break;
			case 1:
				tets[i].M[0] = 10.0;
				tets[i].M[1] = 0.0;
				tets[i].M[2] = 0.0;
				tets[i].M[3] = 10.0;
				tets[i].M[4] = 0.0;
				tets[i].M[5] = 10.0;
				break;


			}
		}


	}
	else
	{
		for(int i =0; i< numtet; i++)
		{
			tets[i].M[0] = 1.0;
			tets[i].M[1] = 0.0;
			tets[i].M[2] = 0.0;
			tets[i].M[3] = 1.0;
			tets[i].M[4] = 0.0;
			tets[i].M[5] = 1.0;
		}

	}


	//if(numattr > 0)
	//{
	//	for(int i =0; i< numtet; i++)
	//	{
	//		int mat = (int)attrlist[i];
	//		switch(mat)
	//		{
	//		case 1:
	//			tets[i].speedInv = 1.0;
	//			break;
	//		case 2:
	//			tets[i].speedInv = 1.52; // refractive index of glass 1.52
	//			break;
	//		case 3:
	//			tets[i].speedInv = 1.0;
	//			break;


	//		}
	//	}


	//}
	//else
	//{
	//	for(int i =0; i< numtet; i++)
	//		tets[i].speedInv = 1.0;

	//}


	//for(int i =0; i< numtet; i++)
	//{
	//	tets[i].f[0] = Face(tets[i][0], tets[i][1], tets[i][2]);
	//	tets[i].f[1] = Face(tets[i][1], tets[i][2], tets[i][3]);
	//	tets[i].f[2] = Face(tets[i][2], tets[i][3], tets[i][0]);
	//	tets[i].f[3] = Face(tets[i][3], tets[i][0], tets[i][1]);





	//}


}
// Find the direct neighbors of each vertex
void TetMesh::need_neighbors()
{
	if (!neighbors.empty())
		return;


	cout << "Finding vertex neighbors... " << endl;
	int nv = vertices.size(), nt = tets.size();

	//vector<int> numneighbors(nv);
	//for (int i = 0; i < nt; i++) {
	//	numneighbors[tets[i][0]]++;
	//	numneighbors[tets[i][1]]++;
	//	numneighbors[tets[i][2]]++;
	//	numneighbors[tets[i][3]]++;
	//}

	neighbors.resize(nv);
	//for (int i = 0; i < nv; i++)
	//	neighbors[i].reserve(numneighbors[i]+2); // Slop for boundaries

	for (int i = 0; i < nt; i++) {
		for (int j = 0; j < 4; j++) {
			vector<int> &me = neighbors[tets[i][j]];
			int n1 = tets[i][(j+1)%4];
			int n2 = tets[i][(j+2)%4];
			int n3 = tets[i][(j+3)%4];
			if (find(me.begin(), me.end(), n1) == me.end())
				me.push_back(n1);
			if (find(me.begin(), me.end(), n2) == me.end())
				me.push_back(n2);
			if (find(me.begin(), me.end(), n3) == me.end())
				me.push_back(n3);
		}
	}

	cout << "Done.\n" << endl;
}


// Find the tets touching each vertex
void TetMesh::need_adjacenttets()
{
	if (!adjacenttets.empty())
		return;

	cout << "Finding vertex to triangle maps... " << endl;
	int nv = vertices.size(), nt = tets.size();

	//vector<int> numadjacentfaces(nv);
	//for (int i = 0; i < nt; i++) {
	//	numadjacentfaces[tets[i][0]]++;
	//	numadjacentfaces[tets[i][1]]++;
	//	numadjacentfaces[tets[i][2]]++;
	//}

	adjacenttets.resize(vertices.size());
	//for (int i = 0; i < nv; i++)
	//	adjacentfaces[i].reserve(numadjacentfaces[i]);

	for (int i = 0; i < nt; i++) {
		for (int j = 0; j < 4; j++)
			adjacenttets[tets[i][j]].push_back(i);
	}

	cout << "Done.\n" << endl;
}


void TetMesh::need_speed()
{
}


//void TetMesh::need_oneringstrip()
//{
//	
//
//	int numVert = vertices.size();
//	//oneringstrip.clear();
//	oneringstrips.resize(numVert);
//	oneringspeedI.resize(numVert);
//	for(int i =0; i<numVert; i++)
//	{
//		udword* topology;
//		vector<int> adjtets = adjacenttets[i];
//		//vector<Tet> adjtets = vertOneringTets[i];
//		topology = (udword*)malloc(3*sizeof(udword)*adjtets.size());
//		for(int j=0; j<adjtets.size(); j++)
//		{
//			int tmp=0;
//			for(int k=0; k<4; k++)
//			{
//				
//				if(tets[adjtets[j]][k] != i)
//				//if(adjtets[j][k] != i)
//				{
//					topology[j*3+tmp] = tets[adjtets[j]][k];
//					//topology[j*3+tmp] = adjtets[j][k];
//					tmp++;
//				}
//
//
//			}
//
//		}
//
//		STRIPERCREATE sc;
//		sc.DFaces			= topology;
//		sc.NbFaces			= adjtets.size();
//		sc.AskForWords		= false;
//		sc.ConnectAllStrips	= true;
//		sc.OneSided			= false;
//		sc.SGIAlgorithm		= true;
//
//		Striper Strip;
//		Strip.Init(sc);
//
//		STRIPERRESULT sr;
//		Strip.Compute(sr);
//
//		vector< vector<int> > strips;
//		strips.resize(sr.NbStrips);
//		udword* Refs = (udword*)sr.StripRuns;
//
//		oneringstrips[i].resize( sr.NbStrips);
//		
//		for(udword k=0;k<sr.NbStrips;k++)
//		{
//			
//			udword NbRefs = sr.StripLengths[k];
//			for(udword j=0;j<NbRefs;j++)
//			{
//				strips[k].push_back(*Refs++);
//				//oneringstrip[i][k].push_back(*Refs++);
//			}
//			
//		}
//
//		
//		//printf("vert %d num strips: %d\n",i, sr.NbStrips);
//
//		
//		oneringstrips[i] = strips[0];
//		
//
//
//
//
//
//		free(topology);
//
//
//
//
//
//	}
//
//	
//
//}

bool TetMesh::IsNonObtuse(int v, Tet t)
{
	int D = t.indexof(v);
	int A = (D+1)%4; 
	int B = (D+2)%4; 
	int C = (D+3)%4; 

	point P1 = vertices[t[A]];
	point P2 = vertices[t[B]];
	point P3 = vertices[t[C]];
	point P4 = vertices[t[D]];

	point a = P1 - P4;
	point b = P2 - P4;
	point c = P3 - P4;

	float det = fabs((a ^ (b % c)));

	float al = a.norm();
	float bl = b.norm();
	float cl = c.norm();

	float div = al*bl*cl + (a ^ b)*cl + (a ^ c)*bl + (b ^ c)*al;
	float at = atan2(det, div);
	if(at < 0) at += M_PI; // If det>0 && div<0 atan2 returns < 0, so add pi.
	float omega = 2.0f * at;

	//return omega < M_PI / 2.0;
	return 1;

}


void TetMesh::SplitFace(vector<Tet> &acTets, int v, Tet ct, int nfAdj)
{		
	// get all the four vertices in order
	/* v1         v4
	+-------+
	\     . \
	\   .   \
	\ .     \
	+-------+
	v2         v3 */

	need_neighbors();
	int iV = ct.indexof(v);											// get index of v in terms of cf
	int v1 = v;
	int v2 = ct[(iV+1)%4];
	int v3 = ct[(iV+2)%4];
	int v4 = ct[(iV+3)%4];
	iV = tets[nfAdj].indexof(v2);				// get index of v in terms of adjacent face

	int v5;
	for(int i=0; i<4; i++)
	{
		if(tets[nfAdj][i] != v2 && tets[nfAdj][i] != v3 && tets[nfAdj][i] != v4)
			v5 = tets[nfAdj][i];

	}
	neighbors[v5].push_back(v1);
	//Tet af = tets[nfAdj];

	// create faces (v1,v3,v4) and (v1,v2,v3), check angle at v1
	Tet t1(v1, v2, v3, v5);	
	Tet t2(v1, v3, v4, v5);	
	Tet t3(v1, v2, v4, v5);

	

	if (IsNonObtuse(v,t1))
	{
		acTets.push_back(t1);
	}
	else
	{
		int nfAdj_new = across_face[nfAdj][tets[nfAdj].indexof(v4)];
		if (nfAdj_new > -1)
		{
			SplitFace(acTets,v,t1,nfAdj_new);
			
		}
		else
			printf("NO cross edge!!! Maybe a hole!!\n");
		//SplitFace(acFaces,v,f1,nfAdj_new, currentVert);
	}

	if (IsNonObtuse(v,t2))
	{
		acTets.push_back(t2);
	}
	else
	{
		int nfAdj_new = across_face[nfAdj][tets[nfAdj].indexof(v2)];
		if (nfAdj_new > -1)
		{
			SplitFace(acTets,v,t2,nfAdj_new/*,currentVert*/);
		}
		else
			printf("NO cross edge!!! Maybe a hole!!\n");
		//SplitFace(acFaces,v,f2,nfAdj_new,currentVert);
	}

	if (IsNonObtuse(v,t3))
	{
		acTets.push_back(t3);
	}
	else
	{
		int nfAdj_new = across_face[nfAdj][tets[nfAdj].indexof(v3)];
		if (nfAdj_new > -1)
		{
			SplitFace(acTets,v,t3,nfAdj_new/*,currentVert*/);
		}
		else
			printf("NO cross edge!!! Maybe a hole!!\n");
		//SplitFace(acFaces,v,f2,nfAdj_new,currentVert);
	}
}


void TetMesh::need_across_face()
{
	if (!across_face.empty())
		return;
	need_adjacenttets();

	printf("Finding across-face maps... ");

	int nt = tets.size();
	across_face.resize(nt, Tet(-1,-1,-1, -1));

	for (int i = 0; i < nt; i++) {
		for (int j = 0; j < 4; j++) {
			if (across_face[i][j] != -1)
				continue;
			int v1 = tets[i][(j+1)%4];
			int v2 = tets[i][(j+2)%4];
			int v3 = tets[i][(j+3)%4];
			const vector<int> &a1 = adjacenttets[v1];
			const vector<int> &a2 = adjacenttets[v2];
			const vector<int> &a3 = adjacenttets[v3];
			for (int k1 = 0; k1 < a1.size(); k1++) 
			{
				int other = a1[k1];
				if (other == i)
					continue;
				vector<int>::const_iterator it =
					find(a2.begin(), a2.end(), other);

				vector<int>::const_iterator it2 =
					find(a3.begin(), a3.end(), other);

				if (it == a2.end() || it2 == a3.end())
					continue;

				across_face[i][j] = other;
				break;

				
				
			}
		}
	}

	printf("Done.\n");
}

vector<TetMesh::Tet> TetMesh::GetOneRing(int v)
{
	// make sure we have the across-edge map
	if (across_face.empty())
		need_across_face();

	// variables required
	vector<Tet> oneRingTets;
	vector<Tet> t_tets;

	// get adjacent faces
	int naf = adjacenttets[v].size();

	if (!naf)
	{
		std::cout << "vertex " << v << " has 0 adjacent faces..." << std::endl;
	}
	else
	{
		for (int af = 0; af < naf; af++)
		{
			Tet ct = this->tets[adjacenttets[v][af]];

			t_tets.clear();
			if(IsNonObtuse(v,ct))// check angle: if non-obtuse, return existing face
			{
				//this->colors[cf[0]] = Color::red();
				//this->colors[cf[1]] = Color::red();
				//this->colors[cf[2]] = Color::red();
				//t_tets.push_back(ct);					
			}
			else
			{
				int nfae = this->across_face[adjacenttets[v][af]][ct.indexof(v)];
				if (nfae > -1)
				{
					SplitFace(t_tets,v,ct,nfae/*,currentVert*/);// if obtuse, split face till we get all acute angles
				}
				else
					printf("NO cross edge!!! Maybe a hole!!\n");
				//SplitFace(t_faces,v,cf,nfae,currentVert);// if obtuse, split face till we get all acute angles
			}

			for (int tf = 0; tf < t_tets.size(); tf++)
			{					
				//this->colors[t_faces[tf][0]] = Color::red();
				//this->colors[t_faces[tf][1]] = Color::red();
				//this->colors[t_faces[tf][2]] = Color::red();
				oneRingTets.push_back(t_tets[tf]);
			}
		}				
	}
	//this->colors[v] = Color::green();
	return oneRingTets;
}

void TetMesh::need_oneringtets()
{

	if (vertOneringTets.empty())
	{
		vertOneringTets.resize(vertices.size());
		for (int i=0; i< vertices.size();i++)
		{
			vertOneringTets[i] = GetOneRing(i);

		}
		
	}
	
}

void TetMesh::need_tet_virtual_tets()
{

	
	need_across_face();
	vector<Tet> t_tets;
	Tet t;
	int numTets = tets.size();
	tetVirtualTets.resize(numTets);
	for (int i = 0; i < numTets; i++)
	{
		t_tets.clear();
		t = tets[i];

		for (int j = 0; j< 4 ; j++)
		{
			if(!IsNonObtuse(t[j],t))// check angle: if non-obtuse, return existing face
			{
				t.obtuseV = j;

				int nfae = across_face[i][j];
				if (nfae > -1)
				{
					SplitFace(t_tets,t[j],t,nfae);// if obtuse, split face till we get all acute angles
				}
				else
					printf("NO cross edge!!! Maybe a hole!!\n");
				
			}
		}

		tetVirtualTets[i] = t_tets;
	}
	

	
	
}