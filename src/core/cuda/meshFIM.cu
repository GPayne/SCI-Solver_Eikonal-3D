#include "hip/hip_runtime.h"

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          modified to use only 7 floats for triMem
//1. #define TRIMEMLENGTH   7
//2. in FIMCuda and run_neighbor_check, add initilize old at the begining of iteration
//3. in FIMCuda and run_neighbor_check, s_triMem[tx*TRIMEMLENGTH + 3 + C] = TC after each iteration instead of s_triMem[tx*TRIMEMLENGTH + 6 + C] = TC
//4. in FIMCuda and run_neighbor_check, in the reconcile step, there should be no +3 in fetching the location of triMem
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "meshFIM.h"
#include "Vec.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#ifdef WIN32
#include <io.h>
#define unlink _unlink
#else
#include <unistd.h>
#endif
#include "CUDADefines.h"
#include <time.h>
#include <mycutil.h>
extern "C" {
#include <metis.h>
}


/////declaration for cuda kernels///////////////////////////
extern __global__ void run_reduction(bool *con, bool *blockCon, int* ActiveList, int nActiveBlock, int* blockSizes);


extern __global__ void FIMCuda(float3* d_tetMem0, float3* d_tetMem1, float4* d_tetT, float* d_vertT, float* d_speedInv, int* d_vertMem, int* d_vertMemOutside,
    int* d_BlockSizes, bool* d_con, int* d_ActiveList,
    int m_maxNumInVert, int m_maxVertMappingInside, int m_maxNumOutVertMapping, int nIter);

extern __global__ void CopyOutBack(float4* d_tetT, float* d_vertT, int* d_vertMem, int* d_vertMemOutside, int* d_BlockSizes, int* d_ActiveList, int m_maxNumInVert, int m_maxNumTotalTets, int m_maxVertMappingInside, int m_maxVertMappingOutside);


extern __global__ void run_check_neghbor(float3* d_tetMem0, float3* d_tetMem1, float4* d_tetT, float* d_speedInv, int* d_vertMem, int* d_vertMemOutside,
    int* d_BlockSizes, bool* d_con, int* d_ActiveList,
    int m_maxNumInVert, int m_maxVertMappingInside, int m_maxNumOutVertMapping);







#if __DEVICE_EMULATION__

bool InitCUDA(void)
{
  return true;
}


#else

bool InitCUDA(void)
{
  int count = 0;
  int i = 0;

  hipGetDeviceCount(&count);
  if(count == 0)
  {
    fprintf(stderr, "There is no device.\n");
    return false;
  }

  for(i = 0; i < count; i++)
  {
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop, i) == hipSuccess)
    {
      if(prop.major >= 1)
      {
        break;
      }
    }
  }
  if(i == count)
  {
    fprintf(stderr, "There is no device supporting CUDA.\n");
    return false;
  }
  //hipSetDevice(i);

  hipDeviceProp_t props;
  cudaSafeCall(hipSetDevice(0));

  cudaSafeCall(hipGetDeviceProperties(&props, 0));

  printf("Device 0: \"%s\" with Compute %d.%d capability\n",  props.name, props.major, props.minor);

  printf("CUDA initialized.\n");
  return true;
}

#endif

/////////////////////////////////////////////////////////////////////////////

void meshFIM::GraphPartition_METIS2(int& numBlock, int maxNumBlockVerts) //create .mesh file from trimesh faces and call partnmesh function to partition and create intermediate mesh.npart.N file and then read this file
{

  FILE * outf;

  outf = fopen("tmp.mesh", "w+");
  if(outf == NULL)
  {
    printf("Cannot open mesh file to write!!!!\n");
    exit(1);
  }
  int sz = m_meshPtr->tets.size();

  fprintf(outf, "%d 2\n", sz);

  for(int i = 0; i < sz; i++)
    fprintf(outf, "%d %d %d %d\n", m_meshPtr->tets[i].v[0] + 1, m_meshPtr->tets[i].v[1] + 1, m_meshPtr->tets[i].v[2] + 1, m_meshPtr->tets[i].v[3] + 1);

  fclose(outf);


  // m_meshPtr->tets.clear();

  int numVert = m_meshPtr->vertices.size();

  m_PartitionLabel.resize(numVert);

  char outputFileName[512];

  char meshfile[] = "tmp.mesh";

  if(numBlock == 0)
  {
    numBlock = MAX(numVert / maxNumBlockVerts - 10, 1);


    do
    {
      numBlock++;

      m_BlockSizes.resize(numBlock);
      for(int i = 0; i < numBlock; i++)
      {
        m_BlockSizes[i] = 0;
      }
      partnmesh(meshfile,numBlock);

      sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);


      FILE* partFile = fopen(outputFileName, "r+");
      if(partFile == NULL)
      {
        printf("NO part file found!!!!\n");
        exit(1);
      }


      //int tmp;

      for(int i = 0; i < numVert; i++)
      {
        fscanf(partFile, "%d", &m_PartitionLabel[i]);


      }

      for(int i = 0; i < numVert; i++)
      {


        m_BlockSizes[m_PartitionLabel[i]]++;

      }
      m_maxNumInVert = 0;

      for(int i = 0; i < numBlock; i++)
      {

        m_maxNumInVert = MAX(m_maxNumInVert, m_BlockSizes[i]);
      }

      fclose(partFile);


      sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
      unlink(outputFileName);
      sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
      unlink(outputFileName);

    }
    while(m_maxNumInVert != maxNumBlockVerts);
  }
  else
  {
    m_BlockSizes.resize(numBlock);
    for(int i = 0; i < numBlock; i++)
    {
      m_BlockSizes[i] = 0;
    }

    partnmesh(meshfile, numBlock);

    sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);


    FILE* partFile = fopen(outputFileName, "r+");
    if(partFile == NULL)
    {
      printf("NO part file found!!!!\n");
      exit(1);
    }


    //  int tmp;

    for(int i = 0; i < numVert; i++)
    {
      fscanf(partFile, "%d", &m_PartitionLabel[i]);


    }

    for(int i = 0; i < numVert; i++)
    {


      m_BlockSizes[m_PartitionLabel[i]]++;

    }
    m_maxNumInVert = 0;

    for(int i = 0; i < numBlock; i++)
    {

      m_maxNumInVert = MAX(m_maxNumInVert, m_BlockSizes[i]);
    }

    printf("max num vert is : %d\n", m_maxNumInVert);
    fclose(partFile);


    sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
    unlink(outputFileName);
    sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
    unlink(outputFileName);

  }




  srand((unsigned)time(NULL));

  printf("numBlock is : %d\n", numBlock);






  //m_BlockSizes = new int[numBlock];
  //for(int i =0; i<numBlock; i++)
  // m_BlockSizes[i] = 0;

  m_PartitionInVerts.resize(numBlock);

  for(int i = 0; i < numVert; i++)
  {
    m_PartitionInVerts[m_PartitionLabel[i]].push_back(i);
  }
  unlink("tmp.mesh");
}

void meshFIM::GraphPartition_Square(int squareLength, int squareWidth, int squareHeight, int blockLength, int blockWidth, int blockHeight)
{
  int numVert = m_meshPtr->vertices.size();
  //m_PartitionLabel = new int[numVert];
  m_PartitionLabel.resize(numVert);

  //int numBlockEdge = (squareSize / blockSize);
  //int numBlock = numBlockEdge * numBlockEdge;

  int numBlockLength = (squareLength / blockLength);
  int numBlockWidth = (squareWidth / blockWidth);
  int numBlockHeight = (squareHeight / blockHeight);
  int numBlock = numBlockLength * numBlockWidth*numBlockHeight;


  for(int k = 0; k < squareHeight; k++)
    for(int i = 0; i < squareWidth; i++)
      for(int j = 0; j < squareLength; j++)
      {

        int index = k * squareLength * squareWidth + i * squareLength + j;
        int k2 = k;
        int i2 = i;
        int j2 = j;
        //if(k==squareHeight-1)k2--;
        //if(i==squareHeight-1)i2--;
        //if(j==squareHeight-1)j2--;
        m_PartitionLabel[index] = (k2 / blockHeight) * numBlockLength * numBlockWidth + (i2 / blockWidth) * numBlockLength + (j2 / blockLength);
      }

  m_BlockSizes.resize(numBlock);

  //m_BlockSizes = new int[numBlock];
  for(int i = 0; i < numBlock; i++)
    m_BlockSizes[i] = 0;

  m_PartitionInVerts.resize(numBlock);

  for(int i = 0; i < numVert; i++)
  {
    m_PartitionInVerts[m_PartitionLabel[i]].push_back(i);
    m_BlockSizes[m_PartitionLabel[i]]++;
  }

  m_maxNumInVert = 0;

  for(int i = 0; i < numBlock; i++)
  {
    m_maxNumInVert = MAX(m_maxNumInVert, m_BlockSizes[i]);
  }
  printf("final number of blocks: %d\n", numBlock);
}

void meshFIM::GenerateData(void)
{

  printf("Start GenerateData!\n");

  int numVert = m_meshPtr->vertices.size();

  if(!InitCUDA())
  {
    exit(1);
  }

  float* h_tetMem0;
  float* h_tetMem1;
  float* h_tetT;
  float* h_vertT;
  //float*       h_speedInv;
  int* h_vertMem;
  int* h_vertMemOutside;
  bool* h_blockCon;
  int* h_BlockSizes;
  int* h_BlockLabel;
  vector<int> h_ActiveList;
  vector<int> h_ActiveListNew;

  int* d_ActiveList = 0;
  bool* d_con;
  bool* d_blockCon;
  float3* d_tetMem0;
  float3* d_tetMem1;
  float4* d_tetT;
  float* d_vertT;
  float* d_speedInv;
  int* d_vertMem;
  int* d_vertMemOutside;
  int* d_BlockSizes;

  GetTetMem(h_tetMem0, h_tetMem1, h_tetT);
  GetVertMem(h_vertMem, h_vertMemOutside);
  h_vertT = (float*)malloc(sizeof(float)* m_maxNumInVert * m_numBlock);


  h_blockCon = (bool*)malloc(sizeof(bool) * m_numBlock);
  h_BlockLabel = (int*)malloc(sizeof(int)* m_numBlock);
  h_BlockSizes = (int*)malloc(sizeof(int)* m_numBlock);
  memset(h_blockCon, 1, sizeof(bool) * m_numBlock);
  for(int i = 0; i < m_numBlock; i++) {
    h_BlockLabel[i] = FARP;
    h_BlockSizes[i] = m_BlockSizes[i];
  }

  ////////////////////initialize the seed points for h_tetT//////////////////////////

  printf("Seed size is %lu, source block is %d\n", m_SeedPoints.size(),
      m_PartitionLabel.empty()?-1:
      (m_PartitionLabel[m_SeedPoints.empty()?0:m_SeedPoints[0]]));
  for(int i = 0; i < m_SeedPoints.size(); i++)
  {
    int seed = m_SeedPoints[i];
    int seedBelongToBlock = m_PartitionLabel[seed];
    m_ActiveBlocks.insert(m_ActiveBlocks.end(), seedBelongToBlock);
    h_blockCon[seedBelongToBlock] = false;
    h_BlockLabel[seedBelongToBlock] = ACTIVE;
    // int blockIdx = seedBelongToBlock * m_maxNumTotalFaces * TRIMEMLENGTH;
    for(int j = 0; j < m_blockVertMapping[seed].size(); j++)
    {
      h_tetT[m_blockVertMapping[seed][j]] = 0.0;
    }
  }

  int numActive = m_ActiveBlocks.size();

  printf("Active block number is %d.\n", numActive);


  h_ActiveList.resize(m_numBlock);

  set<int>::iterator activeiter = m_ActiveBlocks.begin();
  for(int i = 0; activeiter != m_ActiveBlocks.end(); activeiter++)
    h_ActiveList[i++] = *activeiter;


  unsigned int timerstart, timerend = 0;


  ///////////////////////malloc GPU memory/////////////////////////////////
  cudaSafeCall((hipMalloc((void**)&d_con, sizeof(bool) * m_numBlock * m_maxNumInVert)));
  cudaSafeCall((hipMalloc((void**)&d_tetMem0, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_tetMem1, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_tetT, sizeof(float)* 4 * m_maxNumTotalTets * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_vertT, sizeof(float)* m_maxNumInVert * m_numBlock)));
  cudaSafeCall( hipMalloc( (void**) &d_speedInv, sizeof(float) * m_maxNumTotalTets * m_numBlock) );
  cudaSafeCall((hipMalloc((void**)&d_vertMem, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingInside)));
  cudaSafeCall((hipMalloc((void**)&d_vertMemOutside, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingOutside)));
  cudaSafeCall((hipMalloc((void**)&d_BlockSizes, sizeof(int)* m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_blockCon, sizeof(bool) * m_numBlock)));
  cudaSafeCall((hipMalloc((void**)&d_ActiveList, sizeof(int)* m_numBlock)));


  //////////////////copy to gpu memories///////////////////////////////
  cudaSafeCall((hipMemcpy(d_tetMem0, h_tetMem0, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_tetMem1, h_tetMem1, sizeof(float)* 3 * m_maxNumTotalTets * m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_tetT, h_tetT, sizeof(float)* 4 * m_maxNumTotalTets * m_numBlock, hipMemcpyHostToDevice)));
  //  cudaSafeCall( hipMemcpy( d_speedInv,h_speedInv, sizeof(float) * m_maxNumTotalTets * m_numBlock , hipMemcpyHostToDevice));
  cudaSafeCall((hipMemcpy(d_vertMem, h_vertMem, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingInside, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_vertMemOutside, h_vertMemOutside, sizeof(int)* m_maxNumInVert * m_numBlock * m_maxVertMappingOutside, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_BlockSizes, h_BlockSizes, sizeof(int)* m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemcpy(d_blockCon, h_blockCon, sizeof(bool) * m_numBlock, hipMemcpyHostToDevice)));
  cudaSafeCall((hipMemset(d_vertT, 0, sizeof(float)* m_maxNumInVert * m_numBlock)));

  int nTotalIter = 0;
  int nIter = 2;


  hipFuncSetCacheConfig(reinterpret_cast<const void*>(FIMCuda), hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(run_check_neghbor), hipFuncCachePreferShared);

  vector< vector<float> > tmp_h_verrT;
  vector< vector<float> > tmp_h_verrT2;
  tmp_h_verrT.resize(m_numBlock);
  tmp_h_verrT2.resize(m_numBlock);

  int totalIterationNumber = 0;
  timerstart = clock();

  printf("Start Iteration!!\n");

  int maxActive = 0;
  while(numActive > 0)
  {
    maxActive = MAX(maxActive, numActive);
    printf("nTotalIter = %d, numActive = %d\n", nTotalIter, numActive);
    ///////step 1: run solver /////////////////////////////////////
    nTotalIter++;
    totalIterationNumber += numActive;
    dim3 dimGrid(numActive, 1);
    //dim3 dimBlock(520, 1);
    dim3 dimBlock(m_maxNumTotalTets, 1);
    cudaSafeCall(hipMemcpy(d_ActiveList, &h_ActiveList[0], sizeof(int)* m_numBlock, hipMemcpyHostToDevice));
    int sharedSize = sizeof(float)* 4 * m_maxNumTotalTets + sizeof(int)* m_maxNumInVert * m_maxVertMappingInside;
    cudaSafeCall((FIMCuda << <dimGrid, dimBlock, sharedSize >> >(d_tetMem0, d_tetMem1, d_tetT, d_vertT, d_speedInv, d_vertMem, d_vertMemOutside,
            d_BlockSizes, d_con, d_ActiveList, m_maxNumInVert, m_maxVertMappingInside, m_maxVertMappingOutside, nIter)));

    dimBlock = dim3(m_maxNumInVert, 1);
    CopyOutBack << <dimGrid, dimBlock >> >(d_tetT, d_vertT, d_vertMem, d_vertMemOutside, d_BlockSizes, d_ActiveList, m_maxNumInVert, m_maxNumTotalTets, m_maxVertMappingInside, m_maxVertMappingOutside);
    //////////////////////step 2: reduction////////////////////////////////////////////////
    dimBlock = dim3(m_maxNumInVert, 1);
    run_reduction << <dimGrid, dimBlock >> >(d_con, d_blockCon, d_ActiveList, numActive, d_BlockSizes);

    //////////////////////////////////////////////////////////////////
    // 3. check neighbor tiles of converged tile
    // Add any active block of neighbor of converged block is inserted
    // to the list
    cudaSafeCall(hipMemcpy(h_blockCon, d_blockCon, m_numBlock * sizeof(bool), hipMemcpyDeviceToHost));

    int nOldActiveBlock = numActive;
    
    numActive = 0;
    
    h_ActiveListNew.clear();

    for(uint i = 0; i < nOldActiveBlock; i++)
    {
      // check neighbors of current active tile
      uint currBlkIdx = h_ActiveList[i];
      
      h_BlockLabel[currBlkIdx] = FARP;
      if(!h_blockCon[currBlkIdx]) // if not converged
      {
        //tmpActive.push_back(currBlkIdx);
        h_BlockLabel[currBlkIdx] = ACTIVE;
        //h_ActiveList[numActive++] = currBlkIdx;
      }
    }

    for(uint i = 0; i < nOldActiveBlock; i++)
    {
      // check neighbors of current active tile
      uint currBlkIdx = h_ActiveList[i];

      if(h_blockCon[currBlkIdx]) //converged
      {
        //h_BlockLabel[currBlkIdx] == FARP;
        set<int> nb = m_BlockNeighbor[currBlkIdx];

        set<int>::iterator iter;
        for(iter = nb.begin(); iter != nb.end(); iter++)
        {
          int currIdx = *iter;

          if(h_BlockLabel[currIdx] == FARP)
            //if(find(h_ActiveListNew.begin(), h_ActiveListNew.end(), currIdx) == h_ActiveListNew.end() && find(tmpActive.begin(), tmpActive.end(), currIdx) == tmpActive.end())
          {
            h_BlockLabel[currIdx] = ACTIVE;
            h_ActiveListNew.push_back(currIdx);
            //m_ActiveBlocks.insert(m_ActiveBlocks.end(), currIdx);
          }
        }
      }
    }

    for(uint i = 0; i < nOldActiveBlock; i++)
    {
      uint currBlkIdx = h_ActiveList[i];
      //h_ActiveListNew[numActiveNew++] = currBlkIdx;
      if(!h_blockCon[currBlkIdx]) // if not converged
      {
        //h_BlockLabel[currBlkIdx] = ACTIVE;
        h_ActiveList[numActive++] = currBlkIdx;
      }
    }
    //      //////////////////////////////////////////////////////////////////
    //      // 4. run solver only once for neighbor blocks of converged block
    //      // current active list contains active blocks and neighbor blocks of
    //      // any converged blocks
    printf("numActiveNew = %lu\n", h_ActiveListNew.size());
    if(h_ActiveListNew.size() > 0)
    {
      int numActiveNew = h_ActiveListNew.size();

      cudaSafeCall(hipMemcpy(d_ActiveList, &h_ActiveListNew[0], numActiveNew * sizeof(int), hipMemcpyHostToDevice));
      dim3 dimGrid(numActiveNew, 1);
      dim3 dimBlock(m_maxNumTotalTets, 1);

      int sharedSize = sizeof(float4) * m_maxNumTotalTets + sizeof(int)* m_maxNumInVert * m_maxVertMappingInside;
      run_check_neghbor << <dimGrid, dimBlock, sharedSize >> >(d_tetMem0, d_tetMem1, d_tetT, d_speedInv, d_vertMem, d_vertMemOutside,
          d_BlockSizes, d_con, d_ActiveList, m_maxNumInVert, m_maxVertMappingInside, m_maxVertMappingOutside);

      ////////////////////////////////////////////////////////////////
      // 5. reduction
      ///////////////////////////////////////////////////////////////
      dimGrid = dim3(numActiveNew, 1);
      dimBlock = dim3(m_maxNumInVert, 1);
      run_reduction << <dimGrid, dimBlock >> >(d_con, d_blockCon, d_ActiveList, numActiveNew, d_BlockSizes);

      //////////////////////////////////////////////////////////////////
      // 6. update active list
      // read back active volume from the device and add
      // active block to active list on the host memory
      cudaSafeCall(hipMemcpy(h_blockCon, d_blockCon, m_numBlock * sizeof(bool), hipMemcpyDeviceToHost));
      for(uint i = 0; i < h_ActiveListNew.size(); i++)
      {

        uint currBlkIdx = h_ActiveListNew[i];
        if(!h_blockCon[currBlkIdx]) // false : activate block (not converged)
        {
          h_ActiveList[numActive++] = currBlkIdx;
        }
        else h_BlockLabel[currBlkIdx] = FARP;
      }
    }
  }
  //
  cudaSafeCall(hipDeviceSynchronize());
  timerend = clock();
  double duration = (double)(timerend - timerstart) / CLOCKS_PER_SEC;

  printf("Computing time : %.10lf s\n",duration);
  //
  cudaSafeCall(hipMemcpy(h_vertT, d_vertT, sizeof(float)* m_maxNumInVert * m_numBlock, hipMemcpyDeviceToHost));

  cudaSafeCall(hipDeviceSynchronize());

  printf("num of max active %d\n", maxActive);

  m_meshPtr->vertT.resize(1);
  m_meshPtr->vertT[0].resize(numVert);

#ifdef _DEBUG

  for(int i = 0; i < m_numBlock; i++)
  {

    tmp_h_verrT[i].resize(m_maxNumInVert);
    for(int j = 0; j < m_maxNumInVert; j++)
    {

      tmp_h_verrT[i][j] = h_vertT[i * m_maxNumInVert + j];


    }
  }

#endif

  //FILE* resultfile = fopen("resultXX.txt", "w+");
  for(int i = 0; i < m_numBlock; i++)
  {
    for(int j = 0; j < m_PartitionInVerts[i].size(); j++)
    {
      //fprintf(resultfile, "%.8f\n", h_vertT[i * m_maxNumInVert + j]);
      m_meshPtr->vertT[0][m_PartitionInVerts[i][j]] = h_vertT[i * m_maxNumInVert + j];
    }
  }
  //fclose(resultfile);

  FILE * resultfile = fopen("result.txt", "w+");
  for(int i = 0; i < numVert; i++)
  {
    fprintf(resultfile, "%.8f\n", m_meshPtr->vertT[0][i]);
  }

  fclose(resultfile);



  printf("The iteration number: %d\n", nTotalIter);
  printf("The total iteration number: %d\n", totalIterationNumber);
  //  printf("The total localsolver calls per vertex: %f\n", totalIterationNumber*m_maxNumTotalFaces*(NITER+1)*3.0 / (float)numVert);
  //
  //  vec_triMem.resize(m_maxNumTotalFaces * numBlock * 3);
  //  float maxVertT = 0;
  //  for(int i = 0 ; i <  m_maxNumTotalFaces * numBlock; i++)
  //  {
  //
  //
  //    vec_triMem[3*i + 0] =  h_triMem[i*TRIMEMLENGTH + 0];
  //    vec_triMem[3*i + 1] =  h_triMem[i*TRIMEMLENGTH + 1];
  //    vec_triMem[3*i + 2] =  h_triMem[i*TRIMEMLENGTH + 2];
  //
  //    if(h_triMem[i*TRIMEMLENGTH + 0] >= LARGENUM_TET)
  //      vec_triMem[3*i + 0] = -2;
  //    if(h_triMem[i*TRIMEMLENGTH + 1] >= LARGENUM_TET)
  //      vec_triMem[3*i + 1] = -2;
  //    if(h_triMem[i*TRIMEMLENGTH + 2] >= LARGENUM_TET)
  //      vec_triMem[3*i + 2] = -2;
  //
  //
  //    maxVertT = MAX(maxVertT,MAX(vec_triMem[3*i + 2] , MAX(vec_triMem[3*i + 1] , vec_triMem[3*i + 0])));
  //
  //  }
  //
  //  int vertIndex = 0;
  //
  //  //for(int i =0; i < numVert; i++)
  //  //{
  //  //
  //  //  m_meshPtr->vertT[i] =  h_triMem[blockVertMapping[i][0]];
  //  //  if(m_meshPtr->vertT[i] == maxVertT)
  //  //    vertIndex = i;
  //
  //
  //  //}
  //
  //
  //  printf("The maximun vertT is: %f, the vert index is: %d \n", maxVertT,vertIndex );
  //  //printf("The vertT 259 is: %f\n", m_meshPtr->vertT[259] );
  //
  //
  //  //
  //  //printf("%s\n", h_vertT);


  cudaSafeCall(hipFree(d_con));
  cudaSafeCall(hipFree(d_blockCon));
  cudaSafeCall(hipFree(d_BlockSizes));
  cudaSafeCall(hipFree(d_speedInv));



  free(h_blockCon);
  free(h_BlockSizes);






}


//void meshFIM::InitializePartition(int numBlock)
//{
//  if (!m_meshPtr)
//  {
//    std::cout << "Label-vector size unknown, please set the mesh first..." << std::endl;
//  }
//  else
//  {
//    // initialize all labels to 'Far'
//    int nv = m_meshPtr->vertices.size();
//    //if (m_VertLabel.size() != nv) m_VertLabel.resize(nv);
//    //if (m_BlockLabel.size() != numBlock) m_BlockLabel.resize(numBlock);
//    //
//    //for (int l = 0; l < nv; l++)
//    //{
//    //  m_VertLabel[l] = LabelType::FarPoint;
//    //}
//
//    //for (int l = 0; l < numBlock; l++)
//    //{
//    //  m_BlockLabel[l] = LabelType::FarPoint;
//    //}
//
//    // if seeed-points are present, treat them differently
//    if (!m_SeedPoints.empty())
//    {
//      for (int s = 0; s < m_SeedPoints.size(); s++)
//      {
//        m_BlockLabel[m_PartitionLabel[m_SeedPoints[s]]] = LabelType::ActivePoint;//m_Label[s] = LabelType::SeedPoint;
//        m_VertLabel[m_SeedPoints[s]] =  LabelType::SeedPoint;
//        m_ActiveBlocks.insert(m_ActiveBlocks.end(), m_PartitionLabel[m_SeedPoints[s]]);
//      }
//    }
//    else
//      cout<< "Initialize seed points before labels!!!" << endl;
//
//
//
//    //m_BlockNeighbor.resize(numBlock);
//    //for(int i=0; i<numBlock; i++)
//    //  for(int j =0; j<m_PartitionInVerts[i].size(); j++)
//    //  {
//    //    vector<int> nbs = m_meshPtr->neighbors[m_PartitionInVerts[i][j]];
//    //    for(int k=0; k<nbs.size(); k++)
//    //    {
//    //      if(m_PartitionLabel[nbs[k]] != i)
//    //        m_BlockNeighbor[i].insert(m_BlockNeighbor[i].end(),m_PartitionLabel[nbs[k]]);
//    //    }
//
//
//
//    //  }
//
//  }
//}

void meshFIM::PartitionTets(int numBlock)
{
  ///////////////////////////////////step 3: partition faces//////////////////////////////////////
  printf("Start PartitionTets ...");
  m_PartitionTets.resize(numBlock);
  m_PartitionNbTets.resize(numBlock);

  int numTets = m_meshPtr->tets.size();
  int numVerts = m_meshPtr->vertices.size();
  TetMesh::Tet t;

  vector<TetMesh::Tet> virtualTets;
  vector<int> virtualTetCnt;

  virtualTetCnt.resize(numBlock);
  m_PartitionVirtualTets.resize(numBlock);
  set<int> labels;

  for(int i = 0; i < numTets; i++)
  {
    t = m_meshPtr->tets[i];
    int vfCnt = m_meshPtr->tetVirtualTets[i].size();


    int obtusevert = t.obtuseV;
    if(obtusevert >= 0)
    {
      virtualTetCnt[m_PartitionLabel[t[obtusevert]]] += vfCnt;
      m_PartitionVirtualTets[m_PartitionLabel[t[obtusevert]]].insert(m_PartitionVirtualTets[m_PartitionLabel[t[obtusevert]]].end(), m_meshPtr->tetVirtualTets[i].begin(), m_meshPtr->tetVirtualTets[i].end());
    }


    labels.clear();
    for(int m = 0; m < 4; m++)
      labels.insert(labels.begin(), m_PartitionLabel[t[m]]);


    if(labels.size() == 1)
    {
      m_PartitionTets[*(labels.begin())].push_back(i);



    }
    else if(labels.size() > 1)
    {
      set<int>::iterator it = labels.begin();
      for(set<int>::iterator it = labels.begin(); it != labels.end(); it++)
      {
        m_PartitionNbTets[*it].push_back(i);
        //for(set<int>::iterator it2 = labels.begin(); it2 != labels.end(); it2++)
        //{
        //  if(*it != *it2)
        //    m_BlockNeighbor[*it].insert(m_BlockNeighbor[*it].end(), *it2);

        //}



      }
    }
    else
      printf("Error!!\n");





  }

  vector<int> PartitionToltalTets;
  PartitionToltalTets.resize(numBlock);
  m_maxNumTotalTets = 0;
  for(int j = 0; j < numBlock; j++)
  {


    PartitionToltalTets[j] = m_PartitionTets[j].size() + m_PartitionNbTets[j].size() + virtualTetCnt[j];
    m_maxNumTotalTets = MAX(PartitionToltalTets[j], m_maxNumTotalTets);
  }

  printf("m_maxNumTotalTets is %d\n", m_maxNumTotalTets);


  //calculate block neighbors.
  m_BlockNeighbor.resize(numBlock);
  for(int i = 0; i < numVerts; i++)
  {
    vector<int> nbs = m_meshPtr->neighbors[i];
    for(int j = 0; j < nbs.size(); j++)
    {
      int nb = nbs[j];
      if(m_PartitionLabel[nb] != m_PartitionLabel[i])
        m_BlockNeighbor[m_PartitionLabel[i]].insert(m_BlockNeighbor[m_PartitionLabel[i]].end(), m_PartitionLabel[nb]);
    }

  }


  printf("done!\n");
}

bool meshFIM::gettetmem(vector<float>& tetmem, TetMesh::Tet t)
{
  bool needswap = false;
  tetmem.resize(6);
  point A = m_meshPtr->vertices[t[0]];
  point B = m_meshPtr->vertices[t[1]];
  point C = m_meshPtr->vertices[t[2]];
  point D = m_meshPtr->vertices[t[3]];

  point AB = B - A;
  point AC = C - A;
  point AD = D - A;


  //float tmp = ( (AB) CROSS (AC) )  DOT (AD);
  //if(tmp< 0)
  //{
  //  needswap = true;
  //  swap(B,C);
  //

  //}


  AC = C - A;
  AD = D - A;
  point BC = C - B;
  point CD = D - C;
  point BD = D - B;

  //float M[] = {1.0, 0.0, 0.0, 4.0, 0.0, 9.0};
  //tetmem[0] = vMv(AC, M, BC);
  //tetmem[1] = vMv(BC, M, CD);
  //tetmem[2] = vMv(AC, M, CD);
  //tetmem[3] = vMv(AD, M, BD);
  //tetmem[4] = vMv(AC, M, AD);
  //tetmem[5] = vMv(BC, M, BD);

  tetmem[0] = vMv(AC, t.M, BC);
  tetmem[1] = vMv(BC, t.M, CD);
  tetmem[2] = vMv(AC, t.M, CD);
  tetmem[3] = vMv(AD, t.M, BD);
  tetmem[4] = vMv(AC, t.M, AD);
  tetmem[5] = vMv(BC, t.M, BD);

  return needswap;

}

void meshFIM::GetTetMem(float* &h_tetMem0, float* &h_tetMem1, float* &h_tetT)
{
  h_tetMem0 = (float*)malloc(3 * sizeof(float)* m_maxNumTotalTets * m_numBlock);
  h_tetMem1 = (float*)malloc(3 * sizeof(float)* m_maxNumTotalTets * m_numBlock);
  h_tetT = (float*)malloc(4 * sizeof(float)* m_maxNumTotalTets * m_numBlock);

  int numTets = m_meshPtr->tets.size();

  int numVert = m_meshPtr->vertices.size();

  m_blockVertMapping.resize(numVert); //for each vertex, store the addresses where it appears in the global triMem array.

  TetMesh::Tet t;


  for(int i = 0; i < m_numBlock; i++)
  {
    int blockIdx = i * m_maxNumTotalTets * 3;
    int numPF = m_PartitionTets[i].size();
    for(int j = 0; j < numPF; j++)
    {

      t = m_meshPtr->tets[m_PartitionTets[i][j]];
      vector<float> tetmem;
      bool needswap = gettetmem(tetmem, t);

      h_tetMem0[blockIdx + j * 3 + 0] = tetmem[0];
      h_tetMem0[blockIdx + j * 3 + 1] = tetmem[1];
      h_tetMem0[blockIdx + j * 3 + 2] = tetmem[2];

      h_tetMem1[blockIdx + j * 3 + 0] = tetmem[3];
      h_tetMem1[blockIdx + j * 3 + 1] = tetmem[4];
      h_tetMem1[blockIdx + j * 3 + 2] = tetmem[5];





      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
      h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;


      m_blockVertMapping[t[0]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 0);
      m_blockVertMapping[t[3]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 3);

      if(needswap)
      {

        m_blockVertMapping[t[1]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
        m_blockVertMapping[t[2]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
      }
      else
      {
        m_blockVertMapping[t[1]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
        m_blockVertMapping[t[2]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);

      }
    }

  }

  for(int i = 0; i < m_numBlock; i++)
  {

    // h_blockCon[i] = 1;

    // h_BlockLabel[i] = m_BlockLabel[i];
    // h_BlockSizes[i] = m_BlockSizes[i];
    int blockIdx = i * m_maxNumTotalTets * 3;



    int numPF = m_PartitionTets[i].size();
    int numPNF = m_PartitionNbTets[i].size();
    int numPVF = m_PartitionVirtualTets[i].size();

    int k = 0;
    int l = 0;

    for(int j = numPF; j < m_maxNumTotalTets; j++)
    {



      if(j < numPF + numPNF)
      {

        vector<float> tetmem;
        t = m_meshPtr->tets[m_PartitionNbTets[i][k]];
        bool needswap = gettetmem(tetmem, t);

        h_tetMem0[blockIdx + j * 3 + 0] = tetmem[0];
        h_tetMem0[blockIdx + j * 3 + 1] = tetmem[1];
        h_tetMem0[blockIdx + j * 3 + 2] = tetmem[2];

        h_tetMem1[blockIdx + j * 3 + 0] = tetmem[3];
        h_tetMem1[blockIdx + j * 3 + 1] = tetmem[4];
        h_tetMem1[blockIdx + j * 3 + 2] = tetmem[5];





        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;



        //       h_speedInv[i * m_maxNumTotalTets + j] = t.speedInv;


        m_blockVertMapping[t[0]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 0);
        m_blockVertMapping[t[3]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 3);
        if(needswap)
        {

          m_blockVertMapping[t[1]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
          m_blockVertMapping[t[2]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
        }
        else
        {
          m_blockVertMapping[t[1]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
          m_blockVertMapping[t[2]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);

        }



        k++;

      }


      else if(j < numPF + numPNF + numPVF)
      {

        t = m_PartitionVirtualTets[i][l];
        vector<float> tetmem;
        bool needswap = gettetmem(tetmem, t);

        h_tetMem0[blockIdx + j * 3 + 0] = tetmem[0];
        h_tetMem0[blockIdx + j * 3 + 1] = tetmem[1];
        h_tetMem0[blockIdx + j * 3 + 2] = tetmem[2];

        h_tetMem1[blockIdx + j * 3 + 0] = tetmem[3];
        h_tetMem1[blockIdx + j * 3 + 1] = tetmem[4];
        h_tetMem1[blockIdx + j * 3 + 2] = tetmem[5];

        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;



        // h_speedInv[i * m_maxNumTotalTets + j] = t.speedInv;

        m_blockVertMapping[t[0]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 0);
        m_blockVertMapping[t[3]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 3);
        if(needswap)
        {

          m_blockVertMapping[t[1]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);
          m_blockVertMapping[t[2]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
        }
        else
        {
          m_blockVertMapping[t[1]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 1);
          m_blockVertMapping[t[2]].push_back(i * m_maxNumTotalTets * 4 + j * 4 + 2);

        }
        l++;
      }
      else
      {
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 0] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 1] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 2] = LARGENUM_TET;
        h_tetT[i * m_maxNumTotalTets * 4 + j * 4 + 3] = LARGENUM_TET;
      }
    }
  }
}

void meshFIM::GetVertMem(int* &h_vertMem, int* &h_vertMemOutside)
{

  int numVert = m_meshPtr->vertices.size();

  m_blockVertMappingInside.resize(numVert);
  m_blockVertMappingOutside.resize(numVert);

  m_maxNumVertMapping = 0;

  for(int i = 0; i < m_numBlock; i++)
  {
    int triIdx = i * TETMEMLENGTH * m_maxNumTotalTets;

    for(int m = 0; m < m_PartitionInVerts[i].size(); m++)
    {

      m_maxNumVertMapping = MAX(m_maxNumVertMapping, m_blockVertMapping[i].size());

      vector<int> tmp = m_blockVertMapping[m_PartitionInVerts[i][m]];


      for(int n = 0; n < tmp.size(); n++)
      {
        if(tmp[n] >= triIdx + 0 && tmp[n] < triIdx + m_maxNumTotalTets * TETMEMLENGTH)
          m_blockVertMappingInside[m_PartitionInVerts[i][m]].push_back(tmp[n]);
        else
        {
          m_blockVertMappingOutside[m_PartitionInVerts[i][m]].push_back(tmp[n]);

        }



      }

    }
  }

  m_maxVertMappingInside = 0;
  m_maxVertMappingOutside = 0;
  for(int i = 0; i < numVert; i++)
  {
    m_maxVertMappingInside = MAX(m_maxVertMappingInside, (m_blockVertMappingInside[i].size()));
    m_maxVertMappingOutside = MAX(m_maxVertMappingOutside, (m_blockVertMappingOutside[i].size()));
  }

  printf("maxVertMappingInside is: %d\n", m_maxVertMappingInside);
  printf("maxVertMappingOutside is: %d\n", m_maxVertMappingOutside);


  h_vertMem = (int*)malloc(sizeof(int)* m_maxVertMappingInside * m_maxNumInVert * m_numBlock);
  for(int i = 0; i < m_numBlock; i++)
  {
    int vertIdx = i * m_maxVertMappingInside * m_maxNumInVert;

    for(int m = 0; m < m_PartitionInVerts[i].size(); m++)
    {

      int tmpsize = m_blockVertMappingInside[m_PartitionInVerts[i][m]].size();

      int n = 0;
      for(; n < tmpsize; n++)
        h_vertMem[vertIdx + m * m_maxVertMappingInside + n] = m_blockVertMappingInside[m_PartitionInVerts[i][m]][n];
      for(; n < m_maxVertMappingInside; n++)
        h_vertMem[vertIdx + m * m_maxVertMappingInside + n] = -1 + i * m_maxNumTotalTets * TETMEMLENGTH;

    }

    for(int m = m_PartitionInVerts[i].size() * m_maxVertMappingInside; m < m_maxNumInVert * m_maxVertMappingInside; m++)
    {
      //h_vertMem[vertIdx + m] = -1;
      h_vertMem[vertIdx + m] = -1 + i * m_maxNumTotalTets*TETMEMLENGTH;
    }
  }


  h_vertMemOutside = (int*)malloc(m_maxNumInVert * m_numBlock * m_maxVertMappingOutside * sizeof(int));

  for(int i = 0; i < m_numBlock; i++)
  {
    int vertIdx = i * m_maxVertMappingOutside * m_maxNumInVert;

    for(int m = 0; m < m_PartitionInVerts[i].size(); m++)
    {

      int tmpsize = m_blockVertMappingOutside[m_PartitionInVerts[i][m]].size();

      int n = 0;
      for(; n < tmpsize; n++)
        h_vertMemOutside[vertIdx + m * m_maxVertMappingOutside + n] = m_blockVertMappingOutside[m_PartitionInVerts[i][m]][n];
      for(; n < m_maxVertMappingOutside; n++)
        h_vertMemOutside[vertIdx + m * m_maxVertMappingOutside + n] = -1;

    }

    for(int m = m_PartitionInVerts[i].size() * m_maxVertMappingOutside; m < m_maxNumInVert * m_maxVertMappingOutside; m++)
    {
      h_vertMemOutside[vertIdx + m] = -1;
    }
  }

  printf("Done GetVertMem!!\n");



}
