#include "hip/hip_runtime.h"

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          modified to use only 7 floats for triMem
//1. #define TRIMEMLENGTH   7
//2. in FIMCuda and run_neighbor_check, add initilize old at the begining of iteration
//3. in FIMCuda and run_neighbor_check, s_triMem[tx*TRIMEMLENGTH + 3 + C] = TC after each iteration instead of s_triMem[tx*TRIMEMLENGTH + 6 + C] = TC
//4. in FIMCuda and run_neighbor_check, in the reconcile step, there should be no +3 in fetching the location of triMem
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "meshFIM2d.h"
#include "Vec.h"
#include <math.h>
#include <metis.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "CUDADefines.h"

#include <time.h>
#ifdef WIN32
#include <io.h>
#define unlink _unlink
#else
#include <unistd.h>
#endif



/////declaration for cuda kernels///////////////////////////
extern __global__ void run_reduction(int *con, int *blockCon,int* ActiveList, int nActiveBlock, int* blockSizes);
extern __global__ void FIMCuda(float* d_triMem,float* d_triMemOut, int* d_vertMem, int* d_vertMemOutside, float* d_edgeMem0,float* d_edgeMem1,float* d_edgeMem2,float* d_speed, int* d_BlockSizes, int* d_con, int* ActiveList, int nActiveBlock,int maxNumTotalFaces, int maxNumVert,/*int nIter, */float m_StopDistance);
extern __global__ void run_check_neighbor(float* d_triMem,float* d_triMemOut, int* d_vertMem,int* d_vertMemOutside,float* d_edgeMem0,float* d_edgeMem1,float* d_edgeMem2, float* d_speed, int* d_BlockSizes, int* d_con,int* d_ActiveList, int numOldActive ,int maxNumTotalFaces, int maxNumVert,int nTotalActive, int m_StopDistance);







#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
  int count = 0;
  int i = 0;

  hipGetDeviceCount(&count);
  if(count == 0) {
    fprintf(stderr, "There is no device.\n");
    return false;
  }

  for(i = 0; i < count; i++) {
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
      if(prop.major >= 1) {
        break;
      }
    }
  }
  if(i == count) {
    fprintf(stderr, "There is no device supporting CUDA.\n");
    return false;
  }
  hipSetDevice(i);

  printf("CUDA initialized.\n");
  return true;
}

#endif

/////////////////////////////////////////////////////////////////////////////

void meshFIM::GraphPartition_METIS(char* partfilename, int numBlock)  //read a metis result .mesh.npart.N file and store into PartitionLabel
{
  int numVert = m_meshPtr->vertices.size();
  //m_PartitionLabel = new int[numVert];
  m_PartitionLabel.resize(numVert);
  FILE* partFile = fopen(partfilename, "r+");
  if(partFile == NULL)
  {
    printf("NO part file found: %s\n", partfilename);
    exit(1);
  }


  for(int i = 0; i < numVert; i++)
  {
    fscanf(partFile, "%d", &m_PartitionLabel[i]);


  }

  srand( (unsigned)time( NULL ) );




  m_BlockSizes.resize(numBlock);

  //m_BlockSizes = new int[numBlock];
  for(int i =0; i<numBlock; i++)
    m_BlockSizes[i] = 0;

  float r,g,b;



  vector< Color > colors;
  colors.resize(numBlock);
  for(int i = 0; i< numBlock; i++)
  {
    r = rand()/(double)RAND_MAX;
    g = rand()/(double)RAND_MAX;
    b = rand()/(double)RAND_MAX;
    colors[i] = Color(r,g,b);
  }
  m_meshPtr->colors.resize(numVert);
  m_PartitionVerts.resize(numBlock);

  for(int i = 0; i<numVert; i++)
  {
    m_PartitionVerts[m_PartitionLabel[i]].push_back(i);


    m_BlockSizes[m_PartitionLabel[i]]++;
    m_meshPtr->colors[i] = colors[m_PartitionLabel[i]];

  }

  m_maxNumVert = 0;

  for(int i = 0 ; i < numBlock; i++)
  {

    m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
  }





}

void meshFIM::partnmesh(char * meshfile, int nparts) {
  int ne, nn, etype, numflag=0, edgecut;
  idxtype *elmnts, *epart, *npart;
  timer IOTmr, DUALTmr;
  char etypestr[4][5] = {"TRI", "TET", "HEX", "QUAD"};

  cleartimer(IOTmr);
  cleartimer(DUALTmr);

  starttimer(IOTmr);
  elmnts = ReadMesh(meshfile, &ne, &nn, &etype);
  stoptimer(IOTmr);

  char str[] = "main: epart";
  char str2[] = "main: npart";
  epart = idxmalloc(ne, str);
  npart = idxmalloc(nn, str2);

  printf("**********************************************************************\n");
  printf("%s", METISTITLE);
  printf("Mesh Information ----------------------------------------------------\n");
  printf("  Name: %s, #Elements: %d, #Nodes: %d, Etype: %s\n\n",
      meshfile, ne, nn, etypestr[etype-1]);
  printf("Partitioning Nodal Graph... -----------------------------------------\n");

  starttimer(DUALTmr);
  METIS_PartMeshNodal(&ne, &nn, elmnts, &etype, &numflag, &nparts, &edgecut, epart, npart);
  stoptimer(DUALTmr);

  printf("  %d-way Edge-Cut: %7d, Balance: %5.2f\n",
      nparts, edgecut, ComputeElementBalance(ne, nparts, epart));

  starttimer(IOTmr);
  WriteMeshPartition(meshfile, nparts, ne, epart, nn, npart);
  stoptimer(IOTmr);


  printf("\nTiming Information --------------------------------------------------\n");
  printf("  I/O:          \t\t %7.3f\n", gettimer(IOTmr));
  printf("  Partitioning: \t\t %7.3f\n", gettimer(DUALTmr));
  //GKfree(&elmnts, &epart, &npart, LTERM);
  free(elmnts);
  free(epart);
  free(npart);
}

void meshFIM::GraphPartition_METIS2(int& numBlock, int maxNumBlockVerts)   //create .mesh file from trimesh faces and call partnmesh method to partition and create intermediate mesh.npart.N file and then read this file
{

  FILE * outf;

  outf = fopen("tmp.mesh", "w+");
  if(outf == NULL)
  {
    printf("Cannot open mesh file to write!!!!\n");
    exit(1);
  }
  int sz = m_meshPtr->faces.size();
  fprintf(outf,"%d 1\n", sz);

  for (int i=0;i<sz;i++)
    fprintf(outf, "%d %d %d\n",m_meshPtr->faces[i].v[0]+1,m_meshPtr->faces[i].v[1]+1,m_meshPtr->faces[i].v[2]+1);
  fclose(outf);

  int numVert = m_meshPtr->vertices.size();

  m_PartitionLabel.resize(numVert);

  char outputFileName[512];

  char meshfile[] = "tmp.mesh";

  if(numBlock == 0)
  {
    numBlock = numVert / maxNumBlockVerts;


    do{
      numBlock++;

      m_BlockSizes.resize(numBlock);
      for(int i=0; i< numBlock;i++)
      {
        m_BlockSizes[i] = 0;
      }
      partnmesh(meshfile,numBlock);

      sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);


      FILE* partFile = fopen(outputFileName, "r+");
      if(partFile == NULL)
      {
        printf("NO part file found: %s\n",outputFileName);
        exit(1);
      }

      for(int i = 0; i < numVert; i++)
      {
        fscanf(partFile, "%d", &m_PartitionLabel[i]);


      }

      for(int i = 0; i<numVert; i++)
      {


        m_BlockSizes[m_PartitionLabel[i]]++;

      }
      m_maxNumVert = 0;

      for(int i = 0 ; i < numBlock; i++)
      {

        m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
      }

      fclose(partFile);


      sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
      unlink(outputFileName);
      sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
      unlink(outputFileName);

    }while(m_maxNumVert != maxNumBlockVerts);
  }
  else
  {
    m_BlockSizes.resize(numBlock);
    for(int i=0; i< numBlock;i++)
    {
      m_BlockSizes[i] = 0;
    }

    partnmesh(meshfile,numBlock);

    sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);

    FILE* partFile = fopen(outputFileName, "r+");
    if(partFile == NULL)
    {
      printf("NO part file found: %s\n",outputFileName);
      exit(1);
    }

    for(int i = 0; i < numVert; i++)
    {
      fscanf(partFile, "%d", &m_PartitionLabel[i]);


    }

    for(int i = 0; i<numVert; i++)
    {


      m_BlockSizes[m_PartitionLabel[i]]++;

    }
    m_maxNumVert = 0;

    for(int i = 0 ; i < numBlock; i++)
    {

      m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
    }

    printf("max num vert is : %d\n", m_maxNumVert);
    fclose(partFile);


    sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
    unlink(outputFileName);
    sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
    unlink(outputFileName);

  }




  srand( (unsigned)time( NULL ) );

  printf("numBlock is : %d\n", numBlock);






  //m_BlockSizes = new int[numBlock];
  //for(int i =0; i<numBlock; i++)
  // m_BlockSizes[i] = 0;

  float r,g,b;



  vector< Color > colors;
  colors.resize(numBlock);
  for(int i = 0; i< numBlock; i++)
  {
    r = rand()/(double)RAND_MAX;
    g = rand()/(double)RAND_MAX;
    b = rand()/(double)RAND_MAX;
    colors[i] = Color(r,g,b);
  }
  m_meshPtr->colors.resize(numVert);
  m_PartitionVerts.resize(numBlock);

  for(int i = 0; i<numVert; i++)
  {
    m_PartitionVerts[m_PartitionLabel[i]].push_back(i);
    m_meshPtr->colors[i] = colors[m_PartitionLabel[i]];

  }

  unlink("tmp.mesh");









}

void meshFIM::GraphPartition_Square(int squareLength,int squareWidth, int blockLength, int blockWidth)
{
  int numVert = m_meshPtr->vertices.size();
  //m_PartitionLabel = new int[numVert];
  m_PartitionLabel.resize(numVert);

  //int numBlockEdge = (squareSize / blockSize);
  //int numBlock = numBlockEdge * numBlockEdge;

  int numBlockLength = (squareLength / blockLength);
  int numBlockWidth  = (squareWidth / blockWidth);
  int numBlock = numBlockLength * numBlockWidth;



  for(int i = 0; i< squareWidth; i++)
    for(int j =0; j< squareLength; j++)
    {
      m_PartitionLabel[i*squareLength+j] = (i/blockWidth) * numBlockLength + (j/blockLength);
    }

  m_BlockSizes.resize(numBlock);

  //m_BlockSizes = new int[numBlock];
  for(int i =0; i<numBlock; i++)
    m_BlockSizes[i] = 0;

  float r,g,b;



  vector< Color > colors;
  colors.resize(numBlock);
  for(int i = 0; i< numBlock; i++)
  {
    r = rand()/(double)RAND_MAX;
    g = rand()/(double)RAND_MAX;
    b = rand()/(double)RAND_MAX;
    colors[i] = Color(r,g,b);
  }
  m_meshPtr->colors.resize(numVert);
  m_PartitionVerts.resize(numBlock);

  for(int i = 0; i<numVert; i++)
  {
    m_PartitionVerts[m_PartitionLabel[i]].push_back(i);


    m_BlockSizes[m_PartitionLabel[i]]++;
    m_meshPtr->colors[i] = colors[m_PartitionLabel[i]];

  }

  m_maxNumVert = 0;

  for(int i = 0 ; i < numBlock; i++)
  {

    m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
  }
  printf("final number of blocks: %d\n", numBlock);





}

void meshFIM::PartitionFaces(int numBlock)
{
  /////////////////////////////////////step 3: partition faces//////////////////////////////////////
  m_PartitionFaces.resize(numBlock);
  m_PartitionNbFaces.resize(numBlock);

  int numFaces = m_meshPtr->faces.size();
  TriMesh::Face f;
  int labelv0;
  int labelv1;
  int labelv2;
  vector<TriMesh::Face> virtualfaces;
  vector<int> virtualFaceCnt;

  virtualFaceCnt.resize(numBlock);
  m_PartitionVirtualFaces.resize(numBlock);

  for(int i = 0; i< numBlock; i++)
    virtualFaceCnt[i] = 0;

  m_BlockNeighbor.resize(numBlock);

  for(int i = 0; i < numFaces; i++)
  {
    f = m_meshPtr->faces[i];
    int vfCnt = m_meshPtr->faceVirtualFaces[i].size();

    for(int k = 0 ; k < 3; k++)
    {
      if(!m_meshPtr->IsNonObtuse(f[k], f))
      {
        virtualFaceCnt[m_PartitionLabel[f[k]]] += vfCnt;
        m_PartitionVirtualFaces[m_PartitionLabel[f[k]]].insert(m_PartitionVirtualFaces[m_PartitionLabel[f[k]]].end(), m_meshPtr->faceVirtualFaces[i].begin(), m_meshPtr->faceVirtualFaces[i].end());
      }

    }





    labelv0 = m_PartitionLabel[f[0]];
    labelv1 = m_PartitionLabel[f[1]];
    labelv2 = m_PartitionLabel[f[2]];

    if(labelv0 == labelv1 && labelv1 == labelv2)
    {
      m_PartitionFaces[labelv0].push_back(i);
      //virtualFaceCnt[labelv0] += vfCnt;
    }
    else if(labelv0 == labelv1 && labelv1 != labelv2)
    {
      m_PartitionNbFaces[labelv0].push_back(i);
      m_PartitionNbFaces[labelv2].push_back(i);

      m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv2);
      m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv0);


      //virtualFaceCnt[labelv0] += vfCnt;
      //virtualFaceCnt[labelv2] += vfCnt;



    }
    else if(labelv0 != labelv1 && labelv1 == labelv2)
    {
      m_PartitionNbFaces[labelv0].push_back(i);
      m_PartitionNbFaces[labelv2].push_back(i);

      m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv2);
      m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv0);

      //virtualFaceCnt[labelv0] += vfCnt;
      //virtualFaceCnt[labelv2] += vfCnt;


    }

    else if(labelv0 == labelv2 && labelv1 != labelv2)
    {
      m_PartitionNbFaces[labelv0].push_back(i);
      m_PartitionNbFaces[labelv1].push_back(i);

      m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv1);
      m_BlockNeighbor[labelv1].insert(m_BlockNeighbor[labelv1].end(), labelv0);

      //virtualFaceCnt[labelv0] += vfCnt;
      //virtualFaceCnt[labelv1] += vfCnt;

    }

    else      //all different
    {
      m_PartitionNbFaces[labelv0].push_back(i);
      m_PartitionNbFaces[labelv1].push_back(i);
      m_PartitionNbFaces[labelv2].push_back(i);

      m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv2);
      m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv0);
      m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv1);
      m_BlockNeighbor[labelv1].insert(m_BlockNeighbor[labelv1].end(), labelv0);
      m_BlockNeighbor[labelv1].insert(m_BlockNeighbor[labelv1].end(), labelv2);
      m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv1);

      //virtualFaceCnt[labelv0] += vfCnt;
      //virtualFaceCnt[labelv1] += vfCnt;
      //virtualFaceCnt[labelv2] += vfCnt;


    }

  }

  vector<int> PartitionToltalFaces;
  PartitionToltalFaces.resize(numBlock);
  m_maxNumTotalFaces = 0;
  for(int j = 0; j < numBlock; j++)
  {


    PartitionToltalFaces[j] = m_PartitionFaces[j].size() + m_PartitionNbFaces[j].size() + virtualFaceCnt[j];
    m_maxNumTotalFaces = MAX(PartitionToltalFaces[j],m_maxNumTotalFaces );
  }
}




void meshFIM::GenerateData(int numBlock)
{

  int numVert = m_meshPtr->vertices.size();
  int numFaces=m_meshPtr->faces.size();





  if(!InitCUDA()) {
    exit(1);
  }

  index       *d_ActiveList= 0;
  int        *d_con;

  int*   d_con_forComputaion;
  int        *d_blockCon;
  float       *d_triMem;
  float      *d_edgeMem0;
  float      *d_edgeMem1;
  float      *d_edgeMem2;
  float*      d_speed;
  float      *d_triMemOut;
  int         *d_vertMem;

  int         *d_BlockSizes;



  index       *h_ActiveList= 0;    //list of active blocks
  int         *h_BlockLabel = 0;   //block active or not
  float       *h_triMem;
  float      *h_edgeMem0;
  float      *h_edgeMem1;
  float      *h_edgeMem2;
  float*      h_speed;
  int         *h_vertMem;
  int         *h_blockCon;

  int         *h_BlockSizes;

  /////////////////////////////malloc cpu memories///////////////////////////
  h_BlockLabel = (int*) malloc(sizeof(int) * numBlock);
  //h_Neighbors = (int*) malloc(sizeof(int) * numVert * MAXNUMNEIGHBOR);
  //h_NonObtuseNeighborFaces = (d_Face*)malloc(sizeof(d_Face) * numVert * MAXNUMNEIGHBOR);

  h_edgeMem0 = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);
  h_edgeMem1 = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);
  h_edgeMem2 = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);
  h_speed    = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);


  h_triMem = (float*)malloc(sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock);
  h_vertMem = (int*)malloc(sizeof(int) * VERTMEMLENGTH * m_maxNumVert * numBlock);
  h_BlockSizes = (int*)malloc(sizeof(int) * numBlock);

  h_blockCon = (int*)malloc(sizeof(int) * numBlock);





  /////////////////////////malloc gpu memories//////////////////////////////

  cudaSafeCall( hipMalloc((void**) &d_con, sizeof(int) * numBlock * REDUCTIONSHARESIZE));

  cudaSafeCall( hipMalloc((void**) &d_con_forComputaion, sizeof(int) * numBlock * REDUCTIONSHARESIZE));

  cudaSafeCall( hipMalloc((void**) &d_blockCon,  sizeof(int) * numBlock));

  cudaSafeCall( hipMalloc((void**) &d_triMem,  sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock));
  cudaSafeCall( hipMalloc((void**) &d_triMemOut,  sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock));
  cudaSafeCall( hipMalloc((void**) &d_edgeMem0,  sizeof(float)  * m_maxNumTotalFaces * numBlock));
  cudaSafeCall( hipMalloc((void**) &d_edgeMem1,  sizeof(float)  * m_maxNumTotalFaces * numBlock));
  cudaSafeCall( hipMalloc((void**) &d_edgeMem2,  sizeof(float)  * m_maxNumTotalFaces * numBlock));

  cudaSafeCall( hipMalloc((void**) &d_speed,  sizeof(float)  * m_maxNumTotalFaces * numBlock));


  //cudaSafeCall( hipMalloc((void**) &d_triMem_forComputation,  sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock));
  cudaSafeCall( hipMalloc((void**) &d_vertMem, sizeof(int) * VERTMEMLENGTH * m_maxNumVert * numBlock));

  cudaSafeCall( hipMalloc((void**) &d_BlockSizes, sizeof(int) * numBlock));
  //cudaSafeCall( hipMalloc((void**) &d_out, sizeof(int) * VERTMEMLENGTH));










  /////////////////initialize cpu memories//////////////////////////////



  vector< vector<int> > blockVertMapping;
  blockVertMapping.resize(numVert);     //for each vertex, store the addresses where it appears in the global triMem array.


  for( int i = 0; i <  numBlock; i++)
  {
    int blockIdx = i * m_maxNumTotalFaces * TRIMEMLENGTH;
    int numPF = m_PartitionFaces[i].size();
    for(int j = 0; j< numPF; j++)
    {

      // printf("%d %f\n",m_PartitionFaces[i][j], m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[0]);
      //h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[0];
      //h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[1];
      //h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[2];

      h_edgeMem0[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[0];
      h_edgeMem1[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[1];
      h_edgeMem2[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[2];


      h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
      h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM;
      h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM;

      h_speed[i * m_maxNumTotalFaces + j]  =  m_meshPtr->faces[m_PartitionFaces[i][j]].speedInv;



      blockVertMapping[m_meshPtr->faces[m_PartitionFaces[i][j]][0]].push_back(blockIdx + j*TRIMEMLENGTH + 0);
      blockVertMapping[m_meshPtr->faces[m_PartitionFaces[i][j]][1]].push_back(blockIdx + j*TRIMEMLENGTH + 1);
      blockVertMapping[m_meshPtr->faces[m_PartitionFaces[i][j]][2]].push_back(blockIdx + j*TRIMEMLENGTH + 2);
    }

  }

  for( int i = 0; i <  numBlock; i++)
  {

    h_blockCon[i] = 1;

    h_BlockLabel[i] = m_BlockLabel[i];
    h_BlockSizes[i] = m_BlockSizes[i];
    int blockIdx = i * m_maxNumTotalFaces * TRIMEMLENGTH;

    int numPF = m_PartitionFaces[i].size();
    int numPNF = m_PartitionNbFaces[i].size();
    int numPVF = m_PartitionVirtualFaces[i].size();

    int k = 0;
    int l = 0;

    for(int j = numPF; j< m_maxNumTotalFaces; j++)
    {



      if( j < numPF + numPNF)
      {
        h_edgeMem0[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionNbFaces[i][k]].edgeLens[0];
        h_edgeMem1[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionNbFaces[i][k]].edgeLens[1];
        h_edgeMem2[i * m_maxNumTotalFaces + j]= m_meshPtr->faces[m_PartitionNbFaces[i][k]].edgeLens[2];


        h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
        h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM;
        h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM;
        h_speed[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionNbFaces[i][k]].speedInv;


        blockVertMapping[m_meshPtr->faces[m_PartitionNbFaces[i][k]][0]].push_back(blockIdx + j*TRIMEMLENGTH + 0);
        blockVertMapping[m_meshPtr->faces[m_PartitionNbFaces[i][k]][1]].push_back(blockIdx + j*TRIMEMLENGTH + 1);
        blockVertMapping[m_meshPtr->faces[m_PartitionNbFaces[i][k]][2]].push_back(blockIdx + j*TRIMEMLENGTH + 2);



        k++;

      }


      else if (j < numPF + numPNF + numPVF)
      {
        h_edgeMem0[i * m_maxNumTotalFaces + j]= m_PartitionVirtualFaces[i][l].edgeLens[0];
        h_edgeMem1[i * m_maxNumTotalFaces + j]= m_PartitionVirtualFaces[i][l].edgeLens[1];
        h_edgeMem2[i * m_maxNumTotalFaces + j] = m_PartitionVirtualFaces[i][l].edgeLens[2];


        h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
        h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM;
        h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM;
        h_speed[i * m_maxNumTotalFaces + j]  =m_PartitionVirtualFaces[i][l].speedInv;



        blockVertMapping[m_PartitionVirtualFaces[i][l][0]].push_back(blockIdx + j*TRIMEMLENGTH + 0);
        blockVertMapping[m_PartitionVirtualFaces[i][l][1]].push_back(blockIdx + j*TRIMEMLENGTH + 1);
        blockVertMapping[m_PartitionVirtualFaces[i][l][2]].push_back(blockIdx + j*TRIMEMLENGTH + 2);

        l++;


      }
      else
      {

        h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
        h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM;
        h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM;
        //h_triMem[blockIdx + j*TRIMEMLENGTH + 3] = LARGENUM;
        //h_triMem[blockIdx + j*TRIMEMLENGTH + 4] = LARGENUM;
        //h_triMem[blockIdx + j*TRIMEMLENGTH + 5] = LARGENUM;
        //h_triMem[blockIdx + j*TRIMEMLENGTH + 6] = LARGENUM;

      }
    }





  }






  m_maxNumVertMapping = 0;
  for(int i =0; i < numVert; i++)
  {
    int blockIndex = m_PartitionLabel[i];
    int tmp = blockVertMapping[i][0];
    int maxi = (blockIndex+1) * m_maxNumTotalFaces * TRIMEMLENGTH;
    int mini = blockIndex * m_maxNumTotalFaces * TRIMEMLENGTH;
    if(  ( tmp< mini) || (tmp >= maxi) )
    {
      for(int j =0; j < blockVertMapping[i].size(); j++)
        if(blockVertMapping[i][j] >= mini && blockVertMapping[i][j] < maxi )
        {
          int swaptmp = tmp;
          blockVertMapping[i][0] = blockVertMapping[i][j];
          blockVertMapping[i][j] = swaptmp;
          break;

        }
    }
    m_maxNumVertMapping = MAX(m_maxNumVertMapping, blockVertMapping[i].size());
  }


  for(int i =0; i < numVert; i++)
  {
    int blockIndex = m_PartitionLabel[i];
    int tmp = blockVertMapping[i][0];
    int maxi = (blockIndex+1) * m_maxNumTotalFaces * TRIMEMLENGTH;
    int mini = blockIndex * m_maxNumTotalFaces * TRIMEMLENGTH;
    if(  ( tmp< mini) || (tmp >= maxi) )
    {
      printf("beyond");
    }
  }



  vector< vector<int> > blockVertMappingInside;
  vector< vector<int> > blockVertMappingOutside;

  blockVertMappingInside.resize(numVert);
  blockVertMappingOutside.resize(numVert);

  for(int i = 0; i< numBlock; i++)
  {
    int triIdx =  i * TRIMEMLENGTH * m_maxNumTotalFaces;

    for(int m  = 0; m < m_PartitionVerts[i].size(); m++)
    {

      vector<int> tmp = blockVertMapping[m_PartitionVerts[i][m]];


      for(int n = 0; n < tmp.size(); n++)
      {
        if( tmp[n] >= triIdx + 0  && tmp[n] < triIdx + m_maxNumTotalFaces*TRIMEMLENGTH)
          blockVertMappingInside[m_PartitionVerts[i][m]].push_back(tmp[n]);
        else
        {
          blockVertMappingOutside[m_PartitionVerts[i][m]].push_back(tmp[n]);

        }



      }

    }
  }

  int maxVertMappingInside = 0;
  int maxVertMappingOutside = 0;
  for(int i =0; i< numVert; i++)
  {
    maxVertMappingInside = MAX(maxVertMappingInside, (blockVertMappingInside[i].size()));
    maxVertMappingOutside = MAX(maxVertMappingInside, (blockVertMappingOutside[i].size()));
  }

  printf("maxVertMappingInside is: %d\n",maxVertMappingInside);
  printf("maxVertMappingOutside is: %d\n",maxVertMappingOutside);


  for(int i = 0; i< numBlock; i++)
  {
    int vertIdx =  i * VERTMEMLENGTH * m_maxNumVert;

    for(int m  = 0; m < m_PartitionVerts[i].size(); m++)
    {

      int tmpsize = blockVertMappingInside[m_PartitionVerts[i][m]].size();

      int n = 0;
      for(; n < tmpsize; n++)
        h_vertMem[vertIdx + m*VERTMEMLENGTH + n] = blockVertMappingInside[m_PartitionVerts[i][m]][n];
      for(;n<VERTMEMLENGTH; n++)

        h_vertMem[vertIdx + m*VERTMEMLENGTH + n] = -1 + i*m_maxNumTotalFaces*TRIMEMLENGTH;

    }

    for(int m = m_PartitionVerts[i].size() * VERTMEMLENGTH; m < m_maxNumVert * VERTMEMLENGTH; m++)
    {
      //h_vertMem[vertIdx + m] = -1;
      h_vertMem[vertIdx + m] = -1 + i*m_maxNumTotalFaces*TRIMEMLENGTH;
    }
  }


  int* h_vertMemOutside = (int*)malloc(m_maxNumVert * numBlock * VERTMEMLENGTHOUTSIDE * sizeof(int));
  int* d_vertMemOutside;
  cudaSafeCall( hipMalloc((void**) &d_vertMemOutside, m_maxNumVert * numBlock * VERTMEMLENGTHOUTSIDE * sizeof(int) ) );

  for(int i = 0; i< numBlock; i++)
  {
    int vertIdx =  i * VERTMEMLENGTHOUTSIDE * m_maxNumVert;

    for(int m  = 0; m < m_PartitionVerts[i].size(); m++)
    {

      int tmpsize = blockVertMappingOutside[m_PartitionVerts[i][m]].size();

      int n = 0;
      for(; n < tmpsize; n++)
        h_vertMemOutside[vertIdx + m*VERTMEMLENGTHOUTSIDE + n] = blockVertMappingOutside[m_PartitionVerts[i][m]][n];
      for(;n<VERTMEMLENGTHOUTSIDE; n++)
        h_vertMemOutside[vertIdx + m*VERTMEMLENGTHOUTSIDE + n] = -1;

    }

    for(int m = m_PartitionVerts[i].size() * VERTMEMLENGTHOUTSIDE; m < m_maxNumVert * VERTMEMLENGTHOUTSIDE; m++)
    {
      h_vertMemOutside[vertIdx + m] = -1;
    }
  }














  h_ActiveList = (int*)malloc(sizeof(int)*numBlock);
  cudaSafeCall( hipMalloc((void**) &d_ActiveList, sizeof(int) * numBlock));

  //////////////////////////////////////////////////////////////////////////////////


  vector<int>  nb;


  int numActive;

  //for(int currentVert = 0; currentVert < 1/*numVert*/; currentVert++)
  //{

  for( int i = 0; i <  numBlock; i++)
  {

    h_blockCon[i] = 1;

    h_BlockLabel[i] = m_BlockLabel[i];
    h_BlockSizes[i] = m_BlockSizes[i];
  }

  //////////////initialize the seed points for h_triMem////////////////////////////////////

  for(int i = 0; i< m_SeedPoints.size(); i++)
  {
    int seed = m_SeedPoints[i];
    int seedBelongToBlock = m_PartitionLabel[seed];
    h_blockCon[seedBelongToBlock] = 0;
    // int blockIdx = seedBelongToBlock * m_maxNumTotalFaces * TRIMEMLENGTH;
    for(int j = 0; j < blockVertMapping[seed].size(); j++)
    {
      h_triMem[blockVertMapping[seed][j]] = 0.0;

    }


  }


  /////////////copy triMem and verMem to a vector just for debugging/////////////////
  vector<float> vec_triMem;
  vector<int>   vec_vertMem;
  vector<int>   vec_vertMemOutside;

  vec_triMem.resize(TRIMEMLENGTH * m_maxNumTotalFaces * numBlock);
  vec_vertMem.resize(VERTMEMLENGTH * m_maxNumVert * numBlock);
  vec_vertMemOutside.resize(VERTMEMLENGTHOUTSIDE * m_maxNumVert * numBlock);
  for(int i =0; i < TRIMEMLENGTH * m_maxNumTotalFaces * numBlock; i++)
    vec_triMem[i] = h_triMem[i];

  for(int i = 0; i< VERTMEMLENGTH * m_maxNumVert * numBlock; i++)
    vec_vertMem[i] = h_vertMem[i];


  for(int i = 0; i< VERTMEMLENGTHOUTSIDE * m_maxNumVert * numBlock; i++)
    vec_vertMemOutside[i] = h_vertMemOutside[i];
  ////////////////////////////////////////////////////////////////////////////

  cudaSafeCall( hipMemcpy( d_triMem,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));

  numActive =m_ActiveBlocks.size();


  set<int>::iterator activeiter = m_ActiveBlocks.begin();
  for(int i =0; activeiter !=  m_ActiveBlocks.end(); activeiter++)
    h_ActiveList[i++] = *activeiter;


  hipEvent_t start, stop, startCopy, stopCopy;
  hipEventCreate(&start);
  hipEventCreate(&startCopy);
  hipEventCreate(&stopCopy);
  hipEventCreate(&stop);
  hipEventRecord(startCopy,0);


  //////////////////copy to gpu memories///////////////////////////////

  cudaSafeCall( hipMemcpy( d_triMem,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_triMemOut,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_edgeMem0,h_edgeMem0, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_edgeMem1,h_edgeMem1, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_edgeMem2,h_edgeMem2, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));

  cudaSafeCall( hipMemcpy( d_speed,h_speed, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));
  //cudaSafeCall( hipMemcpy( d_triMem_forComputation,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_vertMem,h_vertMem, sizeof(int) * m_maxNumVert * numBlock * VERTMEMLENGTH, hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_vertMemOutside,h_vertMemOutside, sizeof(int) * m_maxNumVert * numBlock * VERTMEMLENGTHOUTSIDE, hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_BlockSizes,h_BlockSizes, sizeof(int) * numBlock, hipMemcpyHostToDevice));
  cudaSafeCall( hipMemcpy( d_blockCon,h_blockCon, sizeof(int) * numBlock, hipMemcpyHostToDevice));


  printf("max number of triangles per block: %d\n", m_maxNumTotalFaces);
  int nTotalIter = 0;

  hipEventRecord(start,0);

  int totalIterationNumber = 0;


  while ( numActive > 0)
  {


    ///////////////////////////step 1: run solver //////////////////////////////////////////////////////////////////

    nTotalIter++;

    totalIterationNumber += numActive;
    //printf("number of active block: %d\n", numActive);

    dim3 dimGrid(numActive, 1);
    dim3 dimBlock(m_maxNumTotalFaces, 1);


    cudaSafeCall( hipMemcpy( d_ActiveList,h_ActiveList,sizeof(int) * numBlock, hipMemcpyHostToDevice));

    FIMCuda<<<dimGrid, dimBlock, m_maxNumTotalFaces*TRIMEMLENGTH*sizeof(float)+m_maxNumVert*VERTMEMLENGTH*sizeof(short)>>>( d_triMem,d_triMemOut, d_vertMem,d_vertMemOutside,d_edgeMem0,d_edgeMem1,d_edgeMem2, d_speed, d_BlockSizes, d_con,d_ActiveList, numActive,m_maxNumTotalFaces, m_maxNumVert, m_StopDistance);
    cudaCheckErrors();

    //cudaSafeCall( hipMemcpy(h_triMem, d_triMem,sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH , hipMemcpyDeviceToHost) );
    //vec_triMem.resize(m_maxNumTotalFaces * numBlock * 3);
    //float maxVertT = 0;
    //for(int i = 0 ; i <  m_maxNumTotalFaces * numBlock; i++)
    //{

    //
    //  vec_triMem[3*i + 0] =  h_triMem[i*TRIMEMLENGTH + 3];
    //  vec_triMem[3*i + 1] =  h_triMem[i*TRIMEMLENGTH + 4];
    //  vec_triMem[3*i + 2] =  h_triMem[i*TRIMEMLENGTH + 5];

    //  if(h_triMem[i*TRIMEMLENGTH + 3] >= LARGENUM)
    //    vec_triMem[3*i + 0] = -2;
    //  if(h_triMem[i*TRIMEMLENGTH + 4] >= LARGENUM)
    //    vec_triMem[3*i + 1] = -2;
    //  if(h_triMem[i*TRIMEMLENGTH + 5] >= LARGENUM)
    //    vec_triMem[3*i + 2] = -2;


    //  maxVertT = MAX(maxVertT,MAX(vec_triMem[3*i + 2] , MAX(vec_triMem[3*i + 1] , vec_triMem[3*i + 0])));
    //}

    //cudaSafeCall( hipDeviceSynchronize() );
    //cudaSafeCall( cutStopTimer( timer));
    //printf("FIMCuda Processing time: %f (ms)\n", cutGetTimerValue( timer));



    //////////////////////step 2: reduction////////////////////////////////////////////////


    //timer = 0;
    //cudaSafeCall( cutCreateTimer( &timer));

    //cudaSafeCall( cutStartTimer( timer));

    dimBlock = dim3(REDUCTIONSHARESIZE / 2 , 1);
    run_reduction<<<dimGrid, dimBlock/*, sizeof(int)*m_maxNumVert*/>>>(d_con, d_blockCon,d_ActiveList, numActive, d_BlockSizes);
    cudaCheckErrors();

    //cudaSafeCall( hipDeviceSynchronize() );
    //cudaSafeCall( cutStopTimer( timer));
    //printf("run reduction Processing time: %f (ms)\n", cutGetTimerValue( timer));




    //////////////////////////////////////////////////////////////////
    // 3. check neighbor tiles of converged tile
    // Add any active block of neighbor of converged block is inserted
    // to the list


    //timer = 0;
    //cudaSafeCall( cutCreateTimer( &timer));

    //cudaSafeCall( cutStartTimer( timer));


    cudaSafeCall( hipMemcpy(h_blockCon, d_blockCon, numBlock*sizeof(int), hipMemcpyDeviceToHost) );

    //cudaSafeCall( hipDeviceSynchronize() );
    //cudaSafeCall( cutStopTimer( timer));
    //printf("compute neighbor copy Processing time: %f (ms)\n", cutGetTimerValue( timer));

    int nOldActiveBlock = numActive;

    //timer = 0;
    //cudaSafeCall( cutCreateTimer( &timer));

    //cudaSafeCall( cutStartTimer( timer));

    //vector<int> tmpActiveBlock;

    for(uint i=0; i<nOldActiveBlock; i++)
    {
      // check neighbors of current active tile
      uint currBlkIdx = h_ActiveList[i];

      if(h_blockCon[currBlkIdx]) // not active : converged
      {
        //h_BlockLabel[currBlkIdx] == FARP;
        set<int> nb = m_BlockNeighbor[currBlkIdx];

        set<int>::iterator iter;
        for( iter = nb.begin(); iter != nb.end() ; iter++)
        {
          int currIdx = *iter;

          if(h_BlockLabel[currIdx] == FARP)
          {
            h_BlockLabel[currIdx] = ACTIVE;
            h_ActiveList[numActive++] = currIdx;
            //m_ActiveBlocks.insert(m_ActiveBlocks.end(), currIdx);
          }
        }
      }
      /*        else
                h_ActiveList[numActive++] = currBlkIdx;*/   // if active block is not convergent, add it to active list and computer again next iter. a bug here: if the acitve block happen to be convengent at next iteration, there will be no active block after check_neighbor.

    }






    //cudaSafeCall( hipDeviceSynchronize() );
    //cudaSafeCall( cutStopTimer( timer));
    //printf("compute neighbor Processing time: %f (ms)\n", cutGetTimerValue( timer));


    //////////////////////////////////////////////////////////////////
    // 4. run solver only once for neighbor blocks of converged block
    // current active list contains active blocks and neighbor blocks of
    // any converged blocks
    //


    cudaSafeCall( hipMemcpy(d_ActiveList, h_ActiveList, numActive*sizeof(int), hipMemcpyHostToDevice) );




    //timer = 0;
    //cudaSafeCall( cutCreateTimer( &timer));

    //cudaSafeCall( cutStartTimer( timer));

    dimGrid = dim3(numActive, 1);
    dimBlock = dim3(m_maxNumTotalFaces, 1);


    run_check_neighbor<<< dimGrid, dimBlock, m_maxNumTotalFaces*TRIMEMLENGTH*sizeof(float)+m_maxNumVert*VERTMEMLENGTH*sizeof(short)>>>(d_triMemOut, d_triMem,d_vertMem,d_vertMemOutside, d_edgeMem0,d_edgeMem1,d_edgeMem2,d_speed , d_BlockSizes, d_con,d_ActiveList, nOldActiveBlock ,m_maxNumTotalFaces, m_maxNumVert,numActive, m_StopDistance);
    cudaCheckErrors();


    //cudaSafeCall( hipDeviceSynchronize() );
    //cudaSafeCall( cutStopTimer( timer));
    //printf("check neighbor Processing time: %f (ms)\n", cutGetTimerValue( timer));




    //////////////////////////////////////////////////////////////////
    // 5. reduction




    dimGrid = dim3(numActive, 1);
    dimBlock = dim3(REDUCTIONSHARESIZE / 2 , 1);

    //timer = 0;
    //cudaSafeCall( cutCreateTimer( &timer));
    //cudaSafeCall( cutStartTimer( timer));

    run_reduction<<<dimGrid, dimBlock/*, sizeof(int)*m_maxNumVert*/>>>(d_con, d_blockCon,d_ActiveList,numActive, d_BlockSizes);
    cudaCheckErrors();


    //cudaSafeCall( hipDeviceSynchronize() );
    //cudaSafeCall( cutStopTimer( timer));
    //printf("reduction2 Processing time: %f (ms)\n", cutGetTimerValue( timer));



    //////////////////////////////////////////////////////////////////
    // 6. update active list
    // read back active volume from the device and add
    // active block to active list on the host memory


    //timer = 0;
    //cudaSafeCall( cutCreateTimer( &timer));

    //cudaSafeCall( cutStartTimer( timer));


    numActive = 0;
    //m_ActiveBlocks.clear();

    cudaSafeCall( hipMemcpy(h_blockCon, d_blockCon, numBlock*sizeof(int), hipMemcpyDeviceToHost) );
    for(uint i=0; i<numBlock; i++)
    {
      if(!h_blockCon[i]) // false : activate block (not converged)
      {
        h_BlockLabel[i] = ACTIVE;
        h_ActiveList[numActive++] = i;
        //m_ActiveBlocks.insert(m_ActiveBlocks.end(), i);
        //printf("Block %d added\n", i);
      }
      else h_BlockLabel[i] = FARP;
    }

    //cudaSafeCall( hipDeviceSynchronize() );
    //cudaSafeCall( cutStopTimer( timer));
    //printf("Compute active Processing time: %f (ms)\n", cutGetTimerValue( timer));


  }

  cudaSafeCall( hipDeviceSynchronize() );

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  cudaSafeCall( hipMemcpy(h_triMem, d_triMem,sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH , hipMemcpyDeviceToHost) );

  cudaSafeCall( hipDeviceSynchronize() );

  hipEventRecord(stopCopy,0);
  hipEventSynchronize(stopCopy);

  float totalTime, totalAndCopyTime;
  hipEventElapsedTime(&totalTime, start, stop);
  hipEventElapsedTime(&totalAndCopyTime, startCopy, stopCopy);


  cudaCheckErrors();


  printf("Total Processing time: %f (ms)\n", totalTime);
  printf("Total Processing time and copy time: %f (ms)\n", totalAndCopyTime);
  printf("The iteration number: %d\n", nTotalIter );
  printf("The total iteration number: %d\n", totalIterationNumber );
  printf("The total localsolver calls per vertex: %f\n", totalIterationNumber*m_maxNumTotalFaces*(NITER+1)*3.0 / (float)numVert);

  vec_triMem.resize(m_maxNumTotalFaces * numBlock * 3);
  float maxVertT = 0;
  for(int i = 0 ; i <  m_maxNumTotalFaces * numBlock; i++)
  {


    vec_triMem[3*i + 0] =  h_triMem[i*TRIMEMLENGTH + 0];
    vec_triMem[3*i + 1] =  h_triMem[i*TRIMEMLENGTH + 1];
    vec_triMem[3*i + 2] =  h_triMem[i*TRIMEMLENGTH + 2];

    if(h_triMem[i*TRIMEMLENGTH + 0] >= LARGENUM)
      vec_triMem[3*i + 0] = -2;
    if(h_triMem[i*TRIMEMLENGTH + 1] >= LARGENUM)
      vec_triMem[3*i + 1] = -2;
    if(h_triMem[i*TRIMEMLENGTH + 2] >= LARGENUM)
      vec_triMem[3*i + 2] = -2;


    maxVertT = MAX(maxVertT,MAX(vec_triMem[3*i + 2] , MAX(vec_triMem[3*i + 1] , vec_triMem[3*i + 0])));

  }

  int vertIndex = 0;

  for(int i =0; i < numVert; i++)
  {

    m_meshPtr->vertT[i] =  h_triMem[blockVertMapping[i][0]];
    if(m_meshPtr->vertT[i] == maxVertT)
      vertIndex = i;


  }
  FILE * resultfile = fopen("result2D.txt", "w+");
  for(int i = 0; i < numVert; i++)
  {
    fprintf(resultfile, "%.8f\n", m_meshPtr->vertT[i]);
  }

  fclose(resultfile);





  printf("The maximun vertT is: %f, the vert index is: %d \n", maxVertT,vertIndex );
  //printf("The vertT 259 is: %f\n", m_meshPtr->vertT[259] );


  //
  //printf("%s\n", h_vertT);

  //cudaSafeCall( hipFree(d_Vertices));

  //cudaSafeCall( hipFree(d_vertT));
  //cudaSafeCall( hipFree(d_Faces));
  //cudaSafeCall( hipFree(d_VertLabel));
  cudaSafeCall( hipFree(d_ActiveList));
  cudaSafeCall( hipFree(d_triMem));
  cudaSafeCall( hipFree(d_vertMem));
  cudaSafeCall( hipFree(d_edgeMem0));
  cudaSafeCall( hipFree(d_edgeMem1));
  cudaSafeCall( hipFree(d_edgeMem2));

  cudaSafeCall( hipFree(d_speed));

  cudaSafeCall( hipFree(d_con));

  cudaSafeCall( hipFree(d_blockCon));

  //cudaSafeCall( hipFree(d_Neighbors));

  //free(h_Vertices);
  //free(h_vertT);
  //free(h_Faces);
  free(h_ActiveList);
  //free(h_VertLabel);
  free(h_edgeMem0);
  free(h_edgeMem1);
  free(h_edgeMem2);

  free(h_speed);

  free(h_triMem);
  free(h_vertMem);
  //free(h_Neighbors);
  free(h_BlockLabel);
  free(h_blockCon);
  free(h_BlockSizes);





}
