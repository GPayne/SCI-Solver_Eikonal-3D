#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "TriMesh.h"
#include <cstring>
#include "meshFIM2d.h"
#include <time.h>

/************************************************************************/
/* main                                                           */
/************************************************************************/
int main(int argc, char* argv[]) {
  std::string filename;
  bool verbose = false;
  for(int i = 1;i < argc; i++) {
    if (strcmp(argv[i],"-v") == 0)
      verbose = true;
    else if (strcmp(argv[i],"-i") == 0) {
      if (i+1 >= argc) break;
      filename = std::string(argv[i+1]);
      i++;
    }
  }
  clock_t starttime, endtime;
  if (filename.empty())
    filename = "../example_data/SquareMesh_size16.ply";
  TriMesh *themesh = TriMesh::read(filename.c_str());

  //themesh->need_normals();
  //themesh->need_tstrips();
  themesh->need_bsphere();
  //themesh->need_faceedges();
  //themesh->need_across_edge();

  meshFIM* FIMPtr = new meshFIM;

  starttime = clock ();

  std::vector<int> seedPointList(1,0/*,currentVert*/);

  //int squareLength = 1024;
  //int squareWidth = 1024;
  //int squareBlockLength = 8;
  //int squareBlockWidth  = 8;
  //int numBlockLength = (squareLength / squareBlockLength);
  //int numBlockWidth  = (squareWidth / squareBlockWidth);
  //int numBlock = numBlockLength * numBlockWidth;

  int numBlock = /*0*//*16226*//*18729*//*16624*//*5210*//*2346*//*803*//*14295*//*1600*/10003 /*2557*//*3487*//*1566*//*177*//*950*//*3487*//*1750*//*1150*//*2309*/ /*175*//*2800*/;  //for 59021verts, 950 for 64; for 72202verts, 1150 for 64, 2309 for 32.; for dragon, 1600 for 64,for dragon iso, 3487 for 32; for 98687, 1566 for 64; for dragon.ts, 2557 for 64; for dragon.ts_maxSF0.5, 10003 for 64; for square.1.ply, 14295 for 64;for sphereR40_iso.ply, 803 for 64; 2346 for sphereR60_147237.ply;5210 for square328k; 16624 for square_size1024;18729 for square_1.1m;16226 for sphereR60_1024k_split and 1024k_256split
  int maxNumBlockVerts = 64;

  FIMPtr->SetSeedPoint(seedPointList);
  FIMPtr->SetMesh(themesh);
  FIMPtr->SetStopDistance(50000.0);
  FIMPtr->GraphPartition_METIS2( numBlock, maxNumBlockVerts);
  //FIMPtr->GraphPartition_Square(squareLength,squareWidth,
  //squareBlockLength, squareBlockWidth);

  //FIMPtr->GraphPartition_Simple(4,numBlock);
  //FIMPtr->GraphPartition_METIS("sphere_10968verts.mesh.npart.180", numBlock);

  FIMPtr->PartitionFaces(numBlock);
  FIMPtr->InitializeLabels(numBlock);

  FIMPtr->GenerateData(numBlock);

  //for (int i = 0; i<themesh->vertices.size(); i++)
  //  vertT[currentVert][i] = FIMPtr->m_meshPtr->vertT[i];

  endtime = clock();
  double duration = (double)(endtime - starttime) * 1000/ CLOCKS_PER_SEC;

  if (verbose)
    printf("Computing time : %.10lf ms\n",duration);

  delete FIMPtr;
  return 0;
}
