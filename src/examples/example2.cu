#include "hip/hip_runtime.h"
//-------------------------------------------------------------------
//
//  Copyright (C) 2015
//  Scientific Computing & Imaging Institute
//  University of Utah
//
//  Permission is  hereby  granted, free  of charge, to any person
//  obtaining a copy of this software and associated documentation
//  files  ( the "Software" ),  to  deal in  the  Software without
//  restriction, including  without limitation the rights to  use,
//  copy, modify,  merge, publish, distribute, sublicense,  and/or
//  sell copies of the Software, and to permit persons to whom the
//  Software is  furnished  to do  so,  subject  to  the following
//  conditions:
//
//  The above  copyright notice  and  this permission notice shall
//  be included  in  all copies  or  substantial  portions  of the
//  Software.
//
//  THE SOFTWARE IS  PROVIDED  "AS IS",  WITHOUT  WARRANTY  OF ANY
//  KIND,  EXPRESS OR IMPLIED, INCLUDING  BUT NOT  LIMITED  TO THE
//  WARRANTIES   OF  MERCHANTABILITY,  FITNESS  FOR  A  PARTICULAR
//  PURPOSE AND NONINFRINGEMENT. IN NO EVENT  SHALL THE AUTHORS OR
//  COPYRIGHT HOLDERS  BE  LIABLE FOR  ANY CLAIM, DAMAGES OR OTHER
//  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
//  ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE
//  USE OR OTHER DEALINGS IN THE SOFTWARE.
//-------------------------------------------------------------------
//-------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>
#include "TriMesh.h"
#include "meshFIM2d.h"

/************************************************************************/
/* main                                                           */
/************************************************************************/
int main(int argc, char* argv[]) {
  //Verbose option
  bool verbose = false;
  //input filename (minus extension)
  std::string filename;
  for (int i = 0; i < argc; i++)
    if (strcmp(argv[i],"-v") == 0) {
      verbose = true;
    } else if (strcmp(argv[i],"-i") == 0) {
      if (i+1 >= argc) break;
      filename = std::string(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-h") == 0) {
      printf("Usage: ./Example2 [OPTIONS]\n");
      printf("  -h            Show this help.\n");
      printf("  -v            Verbose output.\n");
      printf("  -i INPUT      Use this triangle mesh \n");
      printf("                [default ../example_data/sphere_4196verts.ply]\n");
      exit(0);
    }
  if (filename.empty())
    filename = "../example_data/sphere_4196verts.ply";
  clock_t starttime, endtime;

  TriMesh *themesh = TriMesh::read(filename.c_str(), verbose);

  meshFIM2d* FIMPtr = new meshFIM2d;

  starttime = clock ();

  std::vector<int> seedPointList(1,0/*,currentVert*/);

  int numBlock = 10003;
  int maxNumBlockVerts = 64;

  FIMPtr->SetSeedPoint(seedPointList);
  FIMPtr->SetMesh(themesh);
  FIMPtr->SetStopDistance(50000.0);
  FIMPtr->GraphPartition_METIS2( numBlock, maxNumBlockVerts, verbose);

  FIMPtr->PartitionFaces(numBlock);
  FIMPtr->InitializeLabels(numBlock);

  std::vector< std::vector< float > > results =
    FIMPtr->GenerateData(numBlock, verbose);

  endtime = clock();
  double duration = (double)(endtime - starttime) * 1000/ CLOCKS_PER_SEC;

  if (verbose)
    printf("Computing time : %.10lf ms\n",duration);

  delete FIMPtr;
  return 0;
}
