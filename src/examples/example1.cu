#include "hip/hip_runtime.h"
//-------------------------------------------------------------------
//
//  Copyright (C) 2015
//  Scientific Computing & Imaging Institute
//  University of Utah
//
//  Permission is  hereby  granted, free  of charge, to any person
//  obtaining a copy of this software and associated documentation
//  files  ( the "Software" ),  to  deal in  the  Software without
//  restriction, including  without limitation the rights to  use,
//  copy, modify,  merge, publish, distribute, sublicense,  and/or
//  sell copies of the Software, and to permit persons to whom the
//  Software is  furnished  to do  so,  subject  to  the following
//  conditions:
//
//  The above  copyright notice  and  this permission notice shall
//  be included  in  all copies  or  substantial  portions  of the
//  Software.
//
//  THE SOFTWARE IS  PROVIDED  "AS IS",  WITHOUT  WARRANTY  OF ANY
//  KIND,  EXPRESS OR IMPLIED, INCLUDING  BUT NOT  LIMITED  TO THE
//  WARRANTIES   OF  MERCHANTABILITY,  FITNESS  FOR  A  PARTICULAR
//  PURPOSE AND NONINFRINGEMENT. IN NO EVENT  SHALL THE AUTHORS OR
//  COPYRIGHT HOLDERS  BE  LIABLE FOR  ANY CLAIM, DAMAGES OR OTHER
//  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
//  ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE
//  USE OR OTHER DEALINGS IN THE SOFTWARE.
//-------------------------------------------------------------------
//-------------------------------------------------------------------

#include <Eikonal.h>

int main(int argc, char *argv[])
{
  //3D options
  Eikonal data(false);
  //input filename (minus extension)
  data.filename_ = "../src/test/test_data/sphere334";
  for (int i = 0; i < argc; i++)
    if (strcmp(argv[i],"-v") == 0) {
      data.verbose_ = true;
    } else if (strcmp(argv[i],"-m") == 0) {
      if (i+1 >= argc) break;
      data.maxIterations_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-b") == 0) {
      if (i+1 >= argc) break;
      data.maxBlocks_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i], "-x") == 0) {
      while (i + 1 < argc && argv[i + 1][0] != '-') {
        float val = atof(argv[++i]);
        data.tetSpeedMtx_.push_back(val);
      }
    } else if (strcmp(argv[i], "-s") == 0) {
      if (i + 1 >= argc) break;
      std::string s = std::string(argv[i + 1]);
      if (s == "CURVATURE") {
        data.speedType_ = CURVATURE;
      } else if (s == "NOISE") {
        data.speedType_ = NOISE;
      } else if (s != "ONE") {
        std::cout << "Unknown speed type: " << s << std::endl;
      }
      i++;
    } else if (strcmp(argv[i],"-i") == 0) {
      if (i+1 >= argc) break;
      data.filename_ = std::string(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-h") == 0) {
      printf("Usage: ./Example1 [OPTIONS]\n");
      printf("  -h              Show this help.\n");
      printf("  -v              Verbose output.\n");
      printf("  -s SPEEDTYPE    Speed type is [ONE], CURVATURE, or NOISE.\n");
      printf("  -i INPUT        Use this triangle mesh \n");
      printf("  -b MAX_BLOCKS   Max # of blocks to use\n");
      printf("  -m MAX_ITER     Max # of iterations before quit\n");
      printf("  -x s1, s2, ...  Speed matrix multipliers from tetmesh attributes.\n");
      exit(0);
    }
  data.solveEikonal();
  //write out the VTK files
  data.writeVTK(false); //true to output values at each iter.
  //we know that the solution should be the euclidean distance from the center.
  std::vector <float> solution;
  for (size_t i = 0; i < data.tetMesh_->vertices.size(); i++) {
    float x = data.tetMesh_->vertices[i][0];
    float y = data.tetMesh_->vertices[i][1];
    float z = data.tetMesh_->vertices[i][2];
    solution.push_back(std::sqrt((0.f - x)*(0.f-x)+(0.f-y)*(0.f-y)+(0.f-z)*(0.f-z)));
  }
  if (data.verbose_)
    data.printErrorGraph(solution);
  return 0;
}

